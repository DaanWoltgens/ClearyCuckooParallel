#include "hip/hip_runtime.h"
#include "int_cu.h"

#ifndef ENTRYINCLUDED
#define ENTRYINCLUDED
#include "TableEntry.h"
#endif


template <class ADD, class REM>
class ClearyCuckooEntry : TableEntry <ADD, REM> {

private:
    int Rindex[2] = { 1, 56 };
    int Hindex[2] = { 57, 62 };
    int Oindex[2] = { 63, 63 };

public:
    __host__ __device__
    ClearyCuckooEntry(REM R, int H, bool O, bool onDevice=true) {
        TableEntry<ADD, REM>::val= 0;
        setR(R, onDevice);
        setH(H, onDevice);
        setO(O, onDevice);
    }

    __host__ __device__
    ClearyCuckooEntry() {
        TableEntry<ADD, REM>::val = 0;
    }


    __host__ __device__
    void exchValue(ClearyCuckooEntry* x) {
        //Atomically set this TableEntry<ADD, REM>::value to the new one
        //printf("\t\tBefore: %" PRIu64 ", %" PRIu64 "\n", TableEntry<ADD, REM>::val, x->getValue());
        uint64_cu old = atomicExch(TableEntry<ADD, REM>::getValPtr(), x->getValue());
        //Return an entry with prev TableEntry<ADD, REM>::val
        x->setValue(old);
        //printf("\t\tAfter: %" PRIu64 ", %" PRIu64 "\n", TableEntry<ADD, REM>::val, x->getValue());
        return;
    }


    __host__ __device__
    void setR(REM x, bool onDevice=true) {
        TableEntry<ADD, REM>::setBits(Rindex[0], Rindex[1], x, onDevice);

    }

    __host__ __device__
    REM getR() {
        return (REM)TableEntry<ADD, REM>::getBits(Rindex[0], Rindex[1]);
    }

    __host__ __device__
    void setH(int x, bool onDevice = true) {
        TableEntry<ADD, REM>::setBits(Hindex[0], Hindex[1], x, onDevice);
    }

    __host__ __device__
    int getH() {
        return (int) TableEntry<ADD, REM>::getBits(Hindex[0], Hindex[1]);
    }

    __host__ __device__
    void setO(bool x, bool onDevice = true) {
        TableEntry<ADD, REM>::setBits(Oindex[0], Oindex[1], x, onDevice);
    }

    __host__ __device__
    bool getO() {
        return (bool)TableEntry<ADD, REM>::getBits(Oindex[0], Oindex[1]);
    }

    __host__ __device__
    void print() {
        printf("%" PRIu64  "\n", TableEntry<ADD, REM>::val);
    }

};
