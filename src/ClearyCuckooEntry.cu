#include "hip/hip_runtime.h"
#ifndef ENTRYINCLUDED
#define ENTRYINCLUDED
#include "TableEntry.h"
#endif


template <class ADD, class REM>
class ClearyCuckooEntry : TableEntry <ADD, REM> {

private:
    int Rindex[2] = { 1, 56 };
    int Hindex[2] = { 57, 62 };
    int Oindex[2] = { 63, 64 };

public:
    __host__ __device__
    ClearyCuckooEntry(ADD R, int H, bool O) {
        val = 0;
        setR(R);
        setH(H);
        setO(O);
    }

    __host__ __device__
    ClearyCuckooEntry() {
        val = 0;
    }


    __host__ __device__
    void exchValue(ClearyCuckooEntry* x) {
        //Atomically set this value to the new one
        uint64_t old = atomicExch(&val, x->getValue());
        //Return an entry with prev val
        x->setValue(old);
        return;
    }


    __host__ __device__
    void setR(REM x) {
        setBits(Rindex[0], Rindex[1], x);
    }

    __host__ __device__
    REM getR() {
        return (REM)getBits(Rindex[0], Rindex[1]);
    }

    __host__ __device__
    void setH(int x) {
        setBits(Hindex[0], Hindex[1], x);
    }

    __host__ __device__
    int getH() {
        return (int) getBits(Hindex[0], Hindex[1]);
    }

    __host__ __device__
    void setO(bool x) {
        setBits(Oindex[0], Oindex[1], x);
    }

    __host__ __device__
    bool getO() {
        return (bool)getBits(Oindex[0], Oindex[1]);
    }

    __host__ __device__
    void print() {
        std::cout << std::bitset<64>(val) << "\n";
    }

};