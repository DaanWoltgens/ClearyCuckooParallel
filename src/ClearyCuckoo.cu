#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <stdlib.h>
#include <math.h>
#include <iterator>
#include <set>
#include <inttypes.h>

#ifndef HASHTABLE
#define HASHTABLE
#include "HashTable.h"
#endif

#ifndef HASHINCLUDED
#define HASHINCLUDED
#include "hashfunctions.cu"
#endif

#include "ClearyCuckooEntry.cu"

class ClearyCuckoo : public HashTable{
    //Allows for easy changing of the types
    using addtype = uint32_t;
    using remtype = uint64_t;
    using hashtype = uint64_t;
    using keytype = uint64_t;
    typedef std::pair<addtype, remtype> keyTuple; 

    private:
        //Constant Vars
        const static int HS = 59;       //HashSize
        int MAXLOOPS = 25;
        int MAXREHASHES = 30;

        //Vars at Construction
        int AS;                    //AdressSize
        int RS;                         //RemainderSize
        int tablesize;
        
        //Hash tables
        ClearyCuckooEntry<addtype, remtype>* T;

        int hashcounter = 0;

        //Hash function ID
        int hn;
        int* hashlist;

        __device__
        keyTuple splitKey(keytype key){
            hashtype mask = ((hashtype) 1 << AS) - 1;
            addtype add = key & mask;
            remtype rem = key >> AS ;
            return std::make_pair(add,rem);
        }

        __device__
        uint64_t reformKey(addtype add, remtype rem){
            rem = rem << AS;
            rem += add;
            return rem;
        }

        __host__ __device__
        int* createHashList(int n) {
            int* list = new int[n];

            for (int i = 0; i < n; i++) {
                int newhash = rand() % 32;
                
                bool alreadyexists = false;
                for (int j = 0; j < i; j++) {
                    if (list[j] == newhash) { alreadyexists = true; }
                }

                if (alreadyexists) {
                    i--;
                }
                else {
                    list[i] = newhash;
                }
            }

            return list;
        }

        __device__
        int getNextHash(int* ls, int curr) {
            for (int i = 0; i < hn; i++) {
                if (ls[i] == curr) {
                    if (i + 1 != hn) {
                        return ls[i + 1];
                    }
                    else {
                        return ls[0];
                    }
                }
            }

            //Default return 0 if hash can't be found
            return ls[0];
        }

        __device__
        bool containsHash(int* ls, int query) {
            for (int i = 0; i < hn; i++) {
                if (ls[i] == query) {
                    return true;
                }
            }
            return false;
        }

        /**
         * Function to label which hash function was used on this value
         **/
        __device__
        bool insertIntoTable(keytype k, ClearyCuckooEntry<addtype, remtype>* T, int depth=0){
            keytype x = k;
            int hash = hashlist[0];

            //If the key is already inserted don't do anything
            if (lookup(k, T)) {
                return false;
            }

            //Start the iteration
            int c = 0;

            while(c < MAXLOOPS){
                //Get the key of k
                hashtype hashed1 = RHASH(hash, x);
                keyTuple split1 = splitKey(hashed1);

                //Store the old value
                remtype temp = T[split1.first].getR();
                bool wasoccupied = T[split1.first].getO();
                int oldhash = T[split1.first].getH();

                //Place new value
                //TODO use atomicCAS
                T[split1.first].setR(split1.second);
                T[split1.first].setO(true);
                T[split1.first].setH(hash);

                //If the first spot was open return
                if(!wasoccupied){
                    return true;
                }

                //Otherwise rebuild the original key
                hashtype h_old = reformKey(split1.first, temp);
                keytype k_old = RHASH_INVERSE(oldhash, h_old);

                //Hash with the opposite hash value
                hash = getNextHash(hashlist, oldhash);
                
                c++;
            }

            if(depth>0){return false;}
            //If MAXLOOPS is reached rehash the whole table
            if(!rehash()){
                //If rehash fails, return
                return false;
            }

            if(insertIntoTable(x, T, depth)){return true;}

            return false;
        };

        __device__
        bool rehash(int depth){
            //Prevent recursion of rehashing
            if(depth >0){return false;}

            hashlist = createHashList(hn);

            //Insert the old values in the new table
            for(int i=0; i<tablesize; i++){

                if (!containsHash(hashlist, T[i].getH())) {
                    //Store the old value
                    remtype temp = T[i].getR();
                    int oldhash = T[i].getH();

                    //Delete Entry
                    T[i] = ClearyCuckooEntry<addtype, remtype>();

                    //Insert
                    hashtype h_old = reformKey(i, temp);
                    keytype k_old = RHASH_INVERSE(oldhash, h_old);
                    insertIntoTable(k_old, T, depth);
                }
            }            
            return true;
        };

        __device__
        bool lookup(uint64_t k, ClearyCuckooEntry<addtype, remtype>* T){
            for (int i = 0; i < 32; i++) {
                uint64_t hashed1 = RHASH(hashlist[i], k);
                keyTuple split1 = splitKey(hashed1);
                if (T[split1.first].getR() == split1.second && T[split1.first].getO()) {
                    return true;
                }
            }
            return false;
        };

    
    public:
        /**
         * Constructor
         */
        ClearyCuckoo(int adressSize, int hashNumber){
            AS = adressSize;
            RS = HS-AS;
            tablesize = (int) pow(2,AS);

            T = new ClearyCuckooEntry<addtype, remtype>[tablesize];

            for(int i=0; i<tablesize; i++){
                T[i] = ClearyCuckooEntry<addtype, remtype>();
            }
            
            hn = hashNumber;
            int* hashlist = createHashList(hn);

        }

        /**
         * Destructor
         */
        ~ClearyCuckoo(){
            delete[] T;

            delete[] hashlist;
        }

        __device__
        bool ClearyCuckoo::insert(keytype k){
            //Succesful Insertion
            if(insertIntoTable(k,T)){
                //Reset the Hash Counter
                hashcounter = 0;
                return true;
            }
            return false;
        };

        __device__
        bool ClearyCuckoo::rehash(){
            //Local counter for number of rehashes
            while(!rehash(0) && hashcounter<MAXREHASHES){
                hashcounter++;
            };
            //If counter tripped return
            if(hashcounter >= MAXREHASHES){
                return false;
            }
            hashcounter++;
            return true;
        }
        
        __device__
        bool ClearyCuckoo::lookup(uint64_t k){
            return lookup(k, T);
        };

        __device__
        void ClearyCuckoo::clear(){
            for(int i=0; i<tablesize; i++){
                T[i] = ClearyCuckooEntry<addtype, remtype>();
            }
        }

        __device__
        int ClearyCuckoo::getSize(){
            return tablesize;
        }

        __device__
        void ClearyCuckoo::print(ClearyCuckooEntry<addtype, remtype>* T){
            printf("-----------------------------------\n");
            printf("|i|r|O[i]|key|label|\n");
            for(int i=0; i<tablesize; i++){
                if(T[i].getO()){
                    remtype rem = T[i].getR();
                    int label = T[i].getH();
                    hashtype h = reformKey(i, rem);
                    keytype k = RHASH_INVERSE(label, h);

                    printf("|%-3i|%-10" PRIu64 "|%-3i|%-10" PRIu64 "|%-4i|\n", i, T[i].getR(), T[i].getO(), k, T[i].getH());
                }
            }
            printf("-----------------------------------\n");
        }

        __device__
        void ClearyCuckoo::print(){
            print(T);
        }

        __device__
        void ClearyCuckoo::debug(){}
        
        void setMaxRehashes(int x){
            MAXREHASHES = x;
        }

        void setMaxLoops(int x){
            MAXLOOPS = x;
        }

};