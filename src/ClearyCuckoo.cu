#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <stdlib.h>
#include <math.h>
#include <iterator>
#include <set>
#include <inttypes.h>
#include <random>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#ifndef HASHTABLE
#define HASHTABLE
#include "HashTable.h"
#endif

#ifndef HASHINCLUDED
#define HASHINCLUDED
#include "hashfunctions.cu"
#endif

#include "ClearyCuckooEntry.cu"

class ClearyCuckoo{

/*
*
*  Global Variables
*
*/

    //Allows for easy changing of the types
    using addtype = uint32_t;
    using remtype = uint64_t;
    using hashtype = uint64_t;
    using keytype = uint64_t;

    private:
        //Constant Vars
        const static int HS = 59;       //HashSize
        int MAXLOOPS = 25;
        int MAXREHASHES = 30;

        //Vars at Construction
        int AS;                    //AdressSize
        int RS;                         //RemainderSize
        int tablesize;
        
        //Hash tables
        ClearyCuckooEntry<addtype, remtype>* T;

        int hashcounter = 0;

        //Hash function ID
        int hn;
        int* hashlist;

        __host__ __device__
        addtype getAdd(keytype key){
            hashtype mask = ((hashtype) 1 << AS) - 1;
            addtype add = key & mask;
            return add;
        }

        __host__ __device__
            remtype getRem(keytype key) {
            remtype rem = key >> AS;
            return rem;
        }

        __host__ __device__
        uint64_t reformKey(addtype add, remtype rem){
            rem = rem << AS;
            rem += add;
            return rem;
        }

        __host__ __device__
        void createHashList(int* list) {
            printf("\tCreating Hashlist\n");
            for (int i = 0; i < hn; i++) {
                list[i] = i;
            }
            return;
        }

        __host__ __device__
        int getNextHash(int* ls, int curr) {
            for (int i = 0; i < hn; i++) {
                if (ls[i] == curr) {
                    if (i + 1 != hn) {
                        return ls[i + 1];
                    }
                    else {
                        return ls[0];
                    }
                }
            }

            //Default return 0 if hash can't be found
            return ls[0];
        }

        __host__ __device__
        bool containsHash(int* ls, int query) {
            for (int i = 0; i < hn; i++) {
                if (ls[i] == query) {
                    return true;
                }
            }
            return false;
        }

        /**
         * Function to label which hash function was used on this value
         **/
        __host__ __device__
        bool insertIntoTable(keytype k, ClearyCuckooEntry<addtype, remtype>* T, int depth=0){
            printf("\tInsertintoTable\n");
            keytype x = k;
            int hash = hashlist[0];

            //If the key is already inserted don't do anything
            if (lookup(k, T)) {
                printf("\tAlready Exists\n");
                return false;
            }

            //Start the iteration
            int c = 0;

            while(c < MAXLOOPS){
                //Get the key of k
                hashtype hashed1 = RHASH(hash, x);
                addtype add = getAdd(hashed1);
                remtype rem = getRem(hashed1);

                //Place new value
                printf("\tPlacing New Value\n");
                ClearyCuckooEntry<addtype, remtype> entry = ClearyCuckooEntry<addtype, remtype>(rem, hash, true, false);
                T[add].exchValue(&entry);

                //Store the old value
                remtype temp = entry.getR();
                bool wasoccupied = entry.getO();
                int oldhash = entry.getH();

                

                //If the first spot was open return
                if(!wasoccupied){
                    return true;
                }

                //Otherwise rebuild the original key
                hashtype h_old = reformKey(add, temp);
                keytype k_old = RHASH_INVERSE(oldhash, h_old);

                //Hash with the opposite hash value
                hash = getNextHash(hashlist, oldhash);
                
                c++;
            }

            if(depth>0){return false;}
            //If MAXLOOPS is reached rehash the whole table
            if(!rehash()){
                //If rehash fails, return
                return false;
            }

            if(insertIntoTable(x, T, depth)){return true;}

            return false;
        };

        __host__ __device__
        bool rehash(int depth){
            //Prevent recursion of rehashing
            if(depth >0){return false;}

            createHashList(hashlist);

            //Insert the old values in the new table
            for(int i=0; i<tablesize; i++){

                if (!containsHash(hashlist, T[i].getH())) {
                    //Store the old value
                    remtype temp = T[i].getR();
                    int oldhash = T[i].getH();

                    //Delete Entry
                    T[i] = ClearyCuckooEntry<addtype, remtype>();

                    //Insert
                    hashtype h_old = reformKey(i, temp);
                    keytype k_old = RHASH_INVERSE(oldhash, h_old);
                    insertIntoTable(k_old, T, depth);
                }
            }            
            return true;
        };

        __host__ __device__
        bool lookup(uint64_t k, ClearyCuckooEntry<addtype, remtype>* T){
            for (int i = 0; i < hn; i++) {
                uint64_t hashed1 = RHASH(hashlist[i], k);
                addtype add = getAdd(hashed1);
                remtype rem = getRem(hashed1);
                if (T[add].getR() == rem && T[add].getO()) {
                    return true;
                }
            }
            return false;
        };

    
    public:
        /**
         * Constructor
         */
        ClearyCuckoo() {}

        ClearyCuckoo(int adressSize, int hashNumber){
            printf("Creating ClearyCuckoo Table\n");
            AS = adressSize;
            RS = HS-AS;
            tablesize = (int) pow(2,AS);

            hn = hashNumber;

            printf("\tAllocating Memory\n");
            hipMallocManaged(&T, tablesize * sizeof(ClearyCuckooEntry<addtype,remtype>));
            hipMallocManaged(&hashlist, hn * sizeof(int));

            printf("\tInitializing Entries\n");
            for(int i=0; i<tablesize; i++){
                new (&T[i]) ClearyCuckooEntry<addtype, remtype>();
            }
            
            createHashList(hashlist);
            printf("\tDone\n");

        }

        /**
         * Destructor
         */
        ~ClearyCuckoo(){
            printf("Destroying Table\n");

            hipFree(T);
            hipFree(hashlist);
        }

        __host__ __device__
        bool ClearyCuckoo::insert(uint64_t k){
            //Succesful Insertion
            printf("\tInserting %" PRIu64 "\n", k);
            if(insertIntoTable(k,T,0)){
                //Reset the Hash Counter
                hashcounter = 0;
                return true;
            }
            return false;
        };

        __host__ __device__
        bool ClearyCuckoo::rehash(){
            //Local counter for number of rehashes
            while(!rehash(0) && hashcounter<MAXREHASHES){
                hashcounter++;
            };
            //If counter tripped return
            if(hashcounter >= MAXREHASHES){
                return false;
            }
            hashcounter++;
            return true;
        }
        
        __host__ __device__
        bool ClearyCuckoo::lookup(uint64_t k){
            return lookup(k, T);
        };

        __host__ __device__
        void ClearyCuckoo::clear(){
            for(int i=0; i<tablesize; i++){
                T[i] = ClearyCuckooEntry<addtype, remtype>();
            }
        }

        __host__ __device__
        int ClearyCuckoo::getSize(){
            return tablesize;
        }

        __host__ __device__
        void ClearyCuckoo::print(ClearyCuckooEntry<addtype, remtype>* T){
            printf("------------------------------------------------------------\n");
            printf("|    i     |     R[i]       | O[i] |      key       |label |\n");
            printf("------------------------------------------------------------\n");
            for(int i=0; i<tablesize; i++){
                if(T[i].getO()){
                    remtype rem = T[i].getR();
                    int label = T[i].getH();
                    hashtype h = reformKey(i, rem);
                    keytype k = RHASH_INVERSE(label, h);

                    printf("|%-10i|%-16" PRIu64 "|%-6i|%-16" PRIu64 "|%-6i|\n", i, T[i].getR(), T[i].getO(), k, T[i].getH());
                }
            }
            printf("------------------------------------------------------------\n");
        }

        __host__ __device__
        void ClearyCuckoo::print(){
            print(T);
        }

        __host__ __device__
        void debug(uint64_t i) {
            //printf("%" PRIu64, T[0]);
            printf("%" PRIu64, i);
        }
        
        void setMaxRehashes(int x){
            MAXREHASHES = x;
        }

        void setMaxLoops(int x){
            MAXLOOPS = x;
        }

};