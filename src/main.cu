#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <algorithm>
#include <fstream>
#include <inttypes.h>

#ifndef HASHTABLE
#define HASHTABLE
#include "HashTable.h"
#endif

#ifndef HASHINCLUDED
#define HASHINCLUDED
#include "Hash.cpp"
#endif

#include "ClearyCuckoo.cu"
#include "Cleary.cu"

/*
 *
 *  Global Variables
 *
 */

std::random_device rd;
std::mt19937_64 e2(rd());
std::mt19937 g(rd());

/*
 *
 *	Helper Functions
 * 
 */

bool contains(uint64_t* arr, uint64_t val, int index) {
    for (int i = 0; i < index; i++) {
        if (val == arr[i]) {
            return true;
        }
    }
    return false;
}

uint64_t* generateTestSet(int size) {
    //Random Number generator
    std::uniform_int_distribution<long long int> dist(0, std::llround(std::pow(2, 58)));

    uint64_t* res;
    hipMallocManaged(&res, size * sizeof(uint64_t));

    for (int n = 0; n < size; ++n) {
        uint64_t rand = dist(e2);
        if (!contains(res, rand, n)) {
            res[n] = rand;
        }
        else {
            //Redo the step
            n--;
        }
    }

    return res;
}

/*
 *
 * Main Functions
 *
 */


__global__
void fillTable(int n, uint64_t* vals)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride) {
        printf("Value %i is %" PRIu64 "\n", i, vals[i]);
    }
}


void Test(int N) {
	//Read File
	uint64_t* vals;

    vals = generateTestSet(N);

	//Create Table 1
	fillTable << <1, 256 >> > (N, vals);

	//Create Table 2

    hipFree(vals);
}

int main(void)
{
    printf("Starting\n");
	Test(512);
	//Benchmark();

    return 0;
}