#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <algorithm>
#include <fstream>
#include <inttypes.h>

#ifndef HASHTABLE
#define HASHTABLE
#include "HashTable.h"
#endif

#ifndef HASHINCLUDED
#define HASHINCLUDED
#include "hashfunctions.cu"
#endif

#include "ClearyCuckoo.cu"
#include "Cleary.cu"

/*
 *
 *  Global Variables
 *
 */

std::random_device rd;
std::mt19937_64 e2(rd());
std::mt19937 g(rd());

/*
 *
 *	Helper Functions
 * 
 */

bool contains(uint64_t* arr, uint64_t val, int index) {
    for (int i = 0; i < index; i++) {
        if (val == arr[i]) {
            return true;
        }
    }
    return false;
}

uint64_t* generateTestSet(int size) {
    //Random Number generator
    std::uniform_int_distribution<long long int> dist(0, std::llround(std::pow(2, 58)));

    uint64_t* res;
    hipMallocManaged(&res, size * sizeof(uint64_t));

    for (int n = 0; n < size; ++n) {
        uint64_t rand = dist(e2);
        if (!contains(res, rand, n)) {
            res[n] = rand;
        }
        else {
            //Redo the step
            n--;
        }
    }

    return res;
}

__host__ __device__
uint64_t reformKey(addtype add, remtype rem, int N) {
    rem = rem << N;
    rem += add;
    return rem;
}

uint64_t* generateCollidingSet(int size, int N) {
    uint64_t* res;
    hipMallocManaged(&res, size * sizeof(uint64_t));

    uint64_t add = 7;

    for (int n = 0; n < (int) size/2; ++n) {
        uint64_t num = reformKey(add, n, N);
        uint64_t nval = RHASH_INVERSE(0, num);
        if (!contains(res, nval, n)) {
            res[n] = nval;
        }
        else {
            //Redo the step
            n--;
        }
    }

    add = 10;

    for (int n = ((int)size / 2); n < size; ++n) {
        uint64_t num = reformKey(add, n, N);
        uint64_t nval = RHASH_INVERSE(0, num);
        if (!contains(res, nval, n)) {
            res[n] = nval;
        }
        else {
            //Redo the step
            n--;
        }
    }

    return res;
}

/*
 *
 * Main Functions
 *
 */

__global__
void fillClearyCuckoo(int N, uint64_t* vals, ClearyCuckoo* H)
{   
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < N; i += stride) {
        H->insert(vals[i]);
        //H->print();
    }
}

__global__
void fillCleary(int N, uint64_t* vals, Cleary* H)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < N; i += stride) {
        H->insert(vals[i]);
    }
}

__global__
void checkClearyCuckoo(int N, uint64_t* vals, ClearyCuckoo* H, bool* res)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < N; i += stride) {
        if (H->lookup(vals[i])) {
            *res = false;
        }
    }
}

__global__
void checkCleary(int N, uint64_t* vals, Cleary* H, bool* res)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < N; i += stride) {
        if (H->lookup(vals[i])) {
            *res = false;
        }
    }
}


void TestFill(int N, int tablesize, uint64_t* vals) {
	//Create Table 1
    ClearyCuckoo* cc;
    hipMallocManaged((void**)&cc, sizeof(ClearyCuckoo));
    new (cc) ClearyCuckoo(tablesize, 16);

    printf("Filling ClearyCuckoo\n");
	fillClearyCuckoo << <1, 256 >> > (N, vals, cc);
    hipDeviceSynchronize();
    printf("Devices Synced\n");
    cc->print();

    //Checking 
    bool* res;
    hipMallocManaged(&res, sizeof(bool));
    *res = true;

    checkClearyCuckoo << <1, 256 >> > (N, vals, cc, res);
    hipDeviceSynchronize();
    printf("Devices Synced\n");
    if (res) {
        printf("All still in the table\n");
    }
    else {
        printf("Vals Missing\n");
    }

	//Create Table 2
    Cleary* c;
    hipMallocManaged((void**)&c, sizeof(Cleary));
    new (c) Cleary(tablesize);

    printf("Filling Cleary\n");
    fillCleary << <1, 256 >> > (N, vals, c);
    hipDeviceSynchronize();
    printf("Devices Synced\n");
    c->print();

    //Checking 
    *res = true;
    checkCleary << <1, 256 >> > (N, vals, c, res);
    hipDeviceSynchronize();
    printf("Devices Synced\n");
    if (res) {
        printf("All still in the table\n");
    }
    else {
        printf("Vals Missing\n");
    }

    //Destroy Vars
    hipFree(vals);
    hipFree(cc);
    hipFree(c);
}


__global__
void lockTestDevice(ClearyEntry<addtype, remtype>* T){
    addtype left = 1;
    addtype right = 4;

    while (true) {
        printf("\tGetting First Lock\n");
        if (!T[left].lock()) {
            printf("\tFirst Lock Failed\n");
                continue;
        }

        printf("\tLeft");
        T[left].print();

        printf("\tGetting Second Lock\n");
        if (!T[right].lock()) {
            printf("\tSecond Lock Failed\n");
                printf("\tAbort Locking\n");
            T[left].unlock();
            printf("\tUnlocked\n");
                continue;
        }

        printf("\tRight");
        T[left].print();

        printf("\t'Insertion\' Succeeded\n");
        T[left].unlock();
        T[right].unlock();
        printf("\tUnlocked\n");

        printf("\tLeft");
        T[left].print();
        printf("\tRight");
        T[left].print();

        return;
    }

}

void lockTest() {
    int tablesize = 256;
    ClearyEntry<addtype, remtype>* T;
    hipMallocManaged(&T, tablesize * sizeof(ClearyEntry<addtype, remtype>));

    printf("\tInitializing Entries\n");
    for (int i = 0; i < tablesize; i++) {
        new (&T[i]) ClearyEntry<addtype, remtype>();
    }

    printf("\tStarting Lock Test\n");
    lockTestDevice << <1, 10 >> > (T);
    hipDeviceSynchronize();

    hipFree(T);
}


int main(void)
{
    int testSize = 150;
    int addressSize = 8;
    
    printf("==============================================================================================================\n");
    printf("                              BASIC TEST                              \n");
    printf("==============================================================================================================\n");
    uint64_t* testset1 = generateTestSet(testSize);
    TestFill(testSize, addressSize, testset1);
    

    printf("==============================================================================================================\n");
    printf("                            COLLISION TEST                            \n");
    printf("==============================================================================================================\n");
    uint64_t* testset2 = generateCollidingSet(testSize, addressSize);
    TestFill(testSize, addressSize, testset2);

    /*
    printf("==============================================================================================================\n");
    printf("                            FILL AND CHECK TEST                            \n");
    printf("==============================================================================================================\n");
    
    int sampleSize = 50;
    for (int i = 0; i < sampleSize; i++) {
        printf("\n\tTEST %i\n", i);
        uint64_t* testset3 = generateCollidingSet((int)pow(2, addressSize), addressSize);
        TestFillAndCheck(testSize, addressSize, testset3);
    }
    */

    //printf("Lock Test\n");
    //lockTest();

    return 0;
}