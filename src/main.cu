#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <algorithm>
#include <fstream>
#include <inttypes.h>
#include <chrono>
#include <vector>

#ifndef HASHTABLE
#define HASHTABLE
#include "HashTable.h"
#endif

#ifndef HASHINCLUDED
#define HASHINCLUDED
#include "hashfunctions.cu"
#endif

#include "ClearyCuckoo.cu"
#include "Cleary.cu"

/*
 *
 *  Global Variables
 *
 */

std::random_device rd;
std::mt19937_64 e2(rd());
std::mt19937 g(rd());

/*
 *
 *	Helper Functions
 * 
 */

bool contains(uint64_t* arr, uint64_t val, int index) {
    for (int i = 0; i < index; i++) {
        if (val == arr[i]) {
            return true;
        }
    }
    return false;
}

uint64_t* generateTestSet(int size) {
    //Random Number generator
    std::uniform_int_distribution<long long int> dist(0, std::llround(std::pow(2, 58)));

    uint64_t* res;
    hipMallocManaged(&res, size * sizeof(uint64_t));

    for (int n = 0; n < size; ++n) {
        uint64_t rand = dist(e2);
        if (!contains(res, rand, n)) {
            res[n] = rand;
        }
        else {
            //Redo the step
            n--;
        }
    }

    return res;
}

__host__ __device__
uint64_t reformKey(addtype add, remtype rem, int N) {
    rem = rem << N;
    rem += add;
    return rem;
}

uint64_t* generateCollidingSet(int size, int N) {
    uint64_t* res;
    hipMallocManaged(&res, size * sizeof(uint64_t));

    uint64_t add = 7;

    for (int n = 0; n < (int) size/2; ++n) {
        uint64_t num = reformKey(add, n, N);
        uint64_t nval = RHASH_INVERSE(0, num);
        if (!contains(res, nval, n)) {
            res[n] = nval;
        }
        else {
            //Redo the step
            n--;
        }
    }

    add = 10;

    for (int n = ((int)size / 2); n < size; ++n) {
        uint64_t num = reformKey(add, n, N);
        uint64_t nval = RHASH_INVERSE(0, num);
        if (!contains(res, nval, n)) {
            res[n] = nval;
        }
        else {
            //Redo the step
            n--;
        }
    }

    return res;
}

template <typename T>
void exportToCSV(std::vector<std::vector<T>>* matrix, std::string name) {
    std::ofstream myfile;
    std::string filename = "../results/benchmark/" + name + ".csv";
    myfile.open(filename);
    if (myfile.is_open()) {
        for (int i = 0; i < matrix->size(); i++) {
            for (int j = 0; j < matrix->at(0).size(); j++) {
                myfile << (*matrix)[i][j] << ",";
            }
            myfile << "\n";
        }
        myfile.close();
    }
    else {
        std::cout << "Failed to open file : \n";
    }
}

template <typename T>
void exportToCSV(std::vector<std::vector<std::vector<T>>>* matrix, std::string name) {
    std::ofstream myfile;
    std::string filename = "../results/benchmark/" + name + ".csv";
    myfile.open(filename);
    if (myfile.is_open()) {
        for (int i = 0; i < matrix->size(); i++) {
            for (int j = 0; j < matrix->at(0).size(); j++) {
                for (int k = 0; k < (matrix->at(0)).at(0).size(); k++) {
                    myfile << i << "," << j << "," << k << "," << ((*matrix)[i][j])[k] << "\n";
                }
            }
        }
        myfile.close();
    }
}

/*
 *
 * Main Functions
 *
 */

__global__
void fillClearyCuckoo(int N, uint64_t* vals, ClearyCuckoo* H, addtype begin=0)
{  
    int index = threadIdx.x;
    int stride = blockDim.x;
    //printf("\t\t\t\tStarting Thread:%i\n", index + begin);
    for (int i = index+begin; i < N+begin; i += stride) {
        //printf("\t\t\t\tCC Index:%i\n", i);
        if (!(H->insert(vals[i]))) {
            //printf("!------------ Insertion Failure ------------!\n");
            break;
        }
    }
}

__global__
void fillClearyCuckoo(int N, uint64_t* vals, ClearyCuckoo* H, addtype* occupancy, int* failFlag)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < N; i += stride) {
        if (failFlag[0]) {
            break;
        }
        if (!(H->insert(vals[i]))) {
            atomicCAS(&(failFlag[0]), 0, 1);
            break;
        }
        atomicAdd(&occupancy[0], 1);
    }
}

__global__
void fillCleary(int N, uint64_t* vals, Cleary* H, addtype begin=0)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index+begin; i < N+begin; i += stride) {
        if (!(H->insert(vals[i]))) {
            //printf("!------------ Insertion Failure ------------!\n");
            break;
        }
    }
}

__global__
void checkClearyCuckoo(int N, uint64_t* vals, ClearyCuckoo* H, bool* res)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < N; i += stride) {
        if (!(H->lookup(vals[i]))) {
            printf("\tSetting Res:Val %" PRIu64 " Missing\n", vals[i]);
            res[0] = false;
        }
    }
}

__global__
void checkCleary(int N, uint64_t* vals, Cleary* H, bool* res)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < N; i += stride) {
        if (!(H->lookup(vals[i]))) {
            printf("\tSetting Res:Val %" PRIu64 " Missing\n", vals[i]);
            res[0] = false;
        }
    }
}


void TestFill(int N, int tablesize, uint64_t* vals) {
    //Init Var
    printf("Making Check Bool\n");
    bool* res;
    hipMallocManaged((void**)&res, sizeof(bool));
    printf("Assigning Value\n");

	//Create Table 1
    ClearyCuckoo* cc;
    hipMallocManaged((void**)&cc, sizeof(ClearyCuckoo));
    new (cc) ClearyCuckoo(tablesize, 16);

    printf("Filling ClearyCuckoo\n");
	fillClearyCuckoo << <1, 1 >> > (N, vals, cc);
    hipDeviceSynchronize();
    printf("Devices Synced\n");
    cc->print();

    //Check Table
    res[0] = true;
    printf("Checking Cleary-Cuckoo\n");
    checkClearyCuckoo << <1, 1 >> > (N, vals, cc, res);
    hipDeviceSynchronize();
    printf("Devices Synced\n");
    if (res[0]) {
        printf("All still in the table\n");
    }
    else {
        printf("!---------------------Vals Missing---------------------!\n");
    }

	//Create Table 2
    Cleary* c;
    hipMallocManaged((void**)&c, sizeof(Cleary));
    new (c) Cleary(tablesize);

    printf("Filling Cleary\n");
    fillCleary << <1, 1 >> > (N, vals, c);
    hipDeviceSynchronize();
    printf("Devices Synced\n");
    c->print();

    //Checking 
    *res = true;
    checkCleary << <1, 1 >> > (N, vals, c, res);
    hipDeviceSynchronize();
    printf("Devices Synced\n");
    if (res[0]) {
        printf("All still in the table\n");
    }
    else {
        printf("!---------------------Vals Missing---------------------!\n");
    }

    //Destroy Vars
    hipFree(res);
    hipFree(cc);
    hipFree(c);
}


__global__
void lockTestDevice(ClearyEntry<addtype, remtype>* T){
    addtype left = 1;
    addtype right = 4;

    while (true) {
        printf("\tGetting First Lock\n");
        if (!T[left].lock()) {
            printf("\tFirst Lock Failed\n");
                continue;
        }

        printf("\tLeft");
        T[left].print();

        printf("\tGetting Second Lock\n");
        if (!T[right].lock()) {
            printf("\tSecond Lock Failed\n");
                printf("\tAbort Locking\n");
            T[left].unlock();
            printf("\tUnlocked\n");
                continue;
        }

        printf("\tRight");
        T[left].print();

        printf("\t'Insertion\' Succeeded\n");
        T[left].unlock();
        T[right].unlock();
        printf("\tUnlocked\n");

        printf("\tLeft");
        T[left].print();
        printf("\tRight");
        T[left].print();

        return;
    }

}

void lockTest() {
    int tablesize = 256;
    ClearyEntry<addtype, remtype>* T;
    hipMallocManaged(&T, tablesize * sizeof(ClearyEntry<addtype, remtype>));

    printf("\tInitializing Entries\n");
    for (int i = 0; i < tablesize; i++) {
        new (&T[i]) ClearyEntry<addtype, remtype>();
    }

    printf("\tStarting Lock Test\n");
    lockTestDevice << <1, 10 >> > (T);
    hipDeviceSynchronize();

    hipFree(T);
}

void entryTest() {
    ClearyEntry<addtype, remtype> c = ClearyEntry<addtype, remtype>();
    c.setR(351629921636382);
    c.print();
    printf("Entry After R %" PRIu64 "\n", c.getR());
}

void Test() {
    const int addressSize = 8;
    const int testSize = std::pow(2, addressSize);
    //const int testSize = 5;
    

    //printf("Lock Test\n");
    //lockTest();

    printf("==============================================================================================================\n");
    printf("                              BASIC TEST                              \n");
    printf("==============================================================================================================\n");
    uint64_t* testset1 = generateTestSet(testSize);
    TestFill(testSize, addressSize, testset1);
    hipFree(testset1);


    printf("==============================================================================================================\n");
    printf("                            COLLISION TEST                            \n");
    printf("==============================================================================================================\n");
    uint64_t* testset2 = generateCollidingSet(testSize, addressSize);
    TestFill(testSize, addressSize, testset2);
    hipFree(testset2);

    printf("\nTESTING DONE\n");
}


/* ================================================================================================================
 *
 *  Benchmark Methods
 * 
 * ================================================================================================================ 
*/

void BenchmarkFilling(int NUM_TABLES, int INTERVAL, int NUM_SAMPLES, int NUM_THREADS, int NUM_LOOPS, int NUM_HASHES) {

    const int WARMUP = 2;

    printf("=====================================================================\n");
    printf("                     Starting INSERTION BENCHMARK                    \n");
    printf("=====================================================================\n");

    std::ofstream myfile;
    std::string filename = "../results/benchmark/benchfill.csv";
    myfile.open(filename);
    if (!myfile.is_open()) {
        printf("File Failed to Open\n");
        return;
    }
    printf("File Opened\n");

    myfile << "tablesize,numthreads,loops,hashes,samples,type,interval,time\n";

    //Tablesizes
    for (int N = 8; N < 8+NUM_TABLES; N++) {
        printf("Table Size:%i\n", N);
        
        int size = std::pow(2, N);
        int setsize = (int)(size / INTERVAL);

        //Number of Threads
        for (int T = 0; T < NUM_THREADS; T++) {
            printf("\tNumber of Threads:%i\n", T);

            for (int L = 0; L < NUM_LOOPS; L++) {
                printf("\t\tNumber of Loops:%i\n", L);

                for (int H = 1; H < NUM_HASHES; H++) {
                    printf("\t\t\tNumber of Hashes:%i\n", H);
                    //Number of samples
                    for (int S = 0; S < NUM_SAMPLES; S++) {
                        printf("\t\t\t\tSample:%i\n", S);
                        uint64_t* vals = generateTestSet(size);

                        //Init Cleary Cuckoo
                        ClearyCuckoo* cc;
                        hipMallocManaged((void**)&cc, sizeof(ClearyCuckoo));
                        new (cc) ClearyCuckoo(N, H);
                        cc->setMaxLoops(L);

                        //Init Cleary
                        Cleary* c;
                        hipMallocManaged((void**)&c, sizeof(Cleary));
                        new (c) Cleary(N);

                        //Loop over intervals

                        for (int j = 0; j < INTERVAL + WARMUP; j++) {
                            //Fill the table
                            //printf("Filling ClearyCuckoo\n");
                            //Start the Timer
                            std::chrono::steady_clock::time_point begin;
                            std::chrono::steady_clock::time_point end;

                            if (j >= WARMUP) {
                                //printf("\t\tBegin: %i End:%i\n", setsize * j, setsize * (j+1));
                                fillClearyCuckoo << <1, std::pow(2, T) >> > (setsize, vals, cc, setsize * (j - WARMUP));
                                hipDeviceSynchronize();
                                //End the timer
                                end = std::chrono::steady_clock::now();

                                myfile << N << "," << std::pow(2, T) << "," << L << "," << H << "," << S << ",cuc," << (j - WARMUP) << "," << std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count() << ",\n";
                            }



                            //Fill the table
                            //printf("Filling Cleary\n");
                            //Start the Timer
                            begin = std::chrono::steady_clock::now();
                            if (j >= WARMUP) {
                                fillCleary << <1, std::pow(2, T) >> > (setsize, vals, c, setsize * (j - WARMUP));
                                hipDeviceSynchronize();
                                //End the timer
                                end = std::chrono::steady_clock::now();

                                myfile << N << "," << std::pow(2, T) << "," << L << "," << H << "," << S << ",cle," << (j - WARMUP) << "," << std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count() << ",\n";
                            }
                        }
                        hipFree(cc);
                        hipFree(c);
                    }
                }
            }
        }
    }

    myfile.close();
    printf("\t\t\tBenchmark Done\n");
}

void BenchmarkMaxOccupancy(int TABLESIZES, int NUM_HASHES, int NUM_LOOPS, int NUM_SAMPLES) {

    printf("=====================================================================\n");
    printf("                   Starting MAX Occupancy Benchmark                  \n");
    printf("=====================================================================\n");

    std::ofstream myfile;
    std::string filename = "../results/benchmark/benchmax.csv";
    myfile.open(filename);
    if (!myfile.is_open()) {
        printf("File Failed to Open\n");
        return;
    }
    printf("File Opened");

    myfile << "tablesize,numhashes,numloops,samples,max\n";

    //MAX_LOOPS
    for (int N = 8; N < 8 + TABLESIZES; N++) {
        printf("Table Size:%i\n", N);
        int size = std::pow(2, N);
        for (int j = 1; j < NUM_HASHES; j++) {
            printf("\tNum of Hashes:%i\n", j);
            for (int k = 0; k < NUM_LOOPS; k++) {
                printf("\t\tNum of Loops:%i\n", k);
                for (int S = 0; S < NUM_SAMPLES; S++) {
                    uint64_t* vals = generateTestSet(size);

                    int* failFlag;
                    hipMallocManaged(&failFlag, sizeof(int));
                    failFlag[0] = false;

                    //Init Cleary Cuckoo
                    ClearyCuckoo* cc;
                    hipMallocManaged((void**)&cc, sizeof(ClearyCuckoo));
                    new (cc) ClearyCuckoo(N, j);
                    cc->setMaxLoops(k);

                    //Var to store num of inserted values
                    addtype* occ;
                    hipMallocManaged(&occ, sizeof(addtype));
                    occ[0] = 0;

                    //Fill the table
                    fillClearyCuckoo << <1, 256 >> > (size, vals, cc, occ, failFlag);
                    hipDeviceSynchronize();

                    myfile << N << "," << j << "," << k << "," << S << "," << occ[0] << ",\n";

                    hipFree(cc);
                    hipFree(occ);
                }
            }
        }

    }

    myfile.close();
    
    printf("\t\t\tStarting MAX Occupancy Benchmark\n");
}


int main(int argc, char* argv[])
{
    if (argc == 1) {
        printf("No Arguments Passed\n");
    }

    if (strcmp(argv[1], "test") == 0) {
        Test();
    }
    else if (strcmp(argv[1], "benchmax") == 0) {
        if (argc < 6) {
            printf("Not Enough Arguments Passed\n");
            printf("Required: TABLESIZES, NUM_HASHES, NUM_LOOPS, NUM_SAMPLES\n");
        }
        BenchmarkMaxOccupancy(std::stoi(argv[2]), std::stoi(argv[3]), std::stoi(argv[4]), std::stoi(argv[5]));
    }
    else if (strcmp(argv[1], "benchfill") == 0) {
        if (argc < 6) {
            printf("Not Enough Arguments Passed\n");
            printf("Required: NUM_TABLES, INTERVAL, NUM_SAMPLES, NUM_THREADS, NUM_LOOPS, NUM_HASHES\n");
        }
        BenchmarkFilling(std::stoi(argv[2]), std::stoi(argv[3]), std::stoi(argv[4]), std::stoi(argv[5]), std::stoi(argv[6]), std::stoi(argv[7]));
    }

    return 0;
}