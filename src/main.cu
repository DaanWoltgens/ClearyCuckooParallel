#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <algorithm>
#include <fstream>
#include <inttypes.h>
#include <chrono>
#include <vector>
#include <string>
#include <iomanip>
#include <sstream>
#include <thread>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "int_cu.h"


#ifndef HASHTABLE
#define HASHTABLE
#include "HashTable.h"
#endif

#ifndef HASHINCLUDED
#define HASHINCLUDED
#include "hashfunctions.cu"
#endif

#include "ClearyCuckoo.cu"
#include "Cleary.cu"

/*
 *
 *  Global Variables
 *
 */

std::random_device rd;
std::mt19937_64 e2(rd());
std::mt19937 g(rd());

/*
 *
 *	Helper Functions
 *
 */

__host__ __device__
addtype getAdd(keytype key, int AS) {
    hashtype mask = ((hashtype)1 << AS) - 1;
    addtype add = key & mask;
    return add;
}

__host__ __device__
remtype getRem(keytype key, int AS) {
    remtype rem = key >> AS;
    return rem;
}

__host__ __device__
bool contains(uint64_cu* arr, uint64_cu val, int index) {
    for (int i = 0; i < index; i++) {
        if (val == arr[i]) {
            return true;
        }
    }
    return false;
}

uint64_cu* generateTestSet(int size) {
    //Random Number generator
    std::uniform_int_distribution<long long int> dist(0, std::llround(std::pow(2, 58)));
    
    #ifdef GPUCODE
    uint64_cu* res;
    gpuErrchk(hipMallocManaged(&res, size * sizeof(uint64_cu)));
    #else
    uint64_cu* res = new uint64_cu[size];
    #endif

    for (int n = 0; n < size; n++) {
        uint64_cu rand = dist(e2);
        if (!contains(res, rand, n)) {

            res[n] = rand;
        }
        else {
            //Redo the step
            n--;
        }
    }
    return res;
}

__global__ 
void setup_kernel(int seed, hiprandState* state) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

__global__
void firstPassGenSet(hiprandState* state, uint64_cu* res, int N, int setsize, int begin) {
    int index = threadIdx.x;
    int stride = blockDim.x;

    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprandState localState = state[idx];

    int maxVal = setsize + begin < N ? setsize + begin : N;

    for (int i = index + begin; i < maxVal; i += stride) {
        float myrandf = hiprand_uniform(&localState);
        uint64_cu newval = myrandf * std::llround(std::pow(2, 58));

        res[i] = newval;
    }
    return;
}

__global__
void secondPassGenSet(hiprandState* state, uint64_cu* res, int N, int setsize, int begin) {
    //printf("Setsize: %i Begin:%i\n", setsize, begin);

    int index = threadIdx.x;
    int stride = blockDim.x;

    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprandState localState = state[idx];

    int maxVal = setsize + begin < N ? setsize + begin : N;
    //printf("MaxVal: %i\n", maxVal);

    for (int i = index + begin; i < maxVal; i += stride) {
        //printf("Index: %i\n", i);
        if (contains(res, res[i], i)) {
            while (true) {
                float myrandf = hiprand_uniform(&localState);
                uint64_cu newval = myrandf * std::llround(std::pow(2, 58));
                //Check if new in table
                if (!contains(res, newval, i)) {
                    res[i] = newval;
                    break;
                }
            }
        }
    }
    return;
}

uint64_cu* generateTestSetParallel(int size, int NUM_THREADS) {

    //Init States
    hiprandState* states;
    gpuErrchk(hipMallocManaged(&states, sizeof(hiprandState) * NUM_THREADS));

    int setsize = 128;
    int split = (int) std::ceil((float)size / (float)setsize);
    split = split == 0 ? 1 : split;

    //Time For Seeding the Randomness
    const auto p1 = std::chrono::system_clock::now();
    setup_kernel << < 1, NUM_THREADS >> > (std::chrono::duration_cast<std::chrono::microseconds>(p1.time_since_epoch()).count(), states);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    //Result array
    uint64_cu* res;
    gpuErrchk(hipMallocManaged(&res, size * sizeof(uint64_cu)));

    //Fill With Values
    for (int i = 0; i < split; i++) {
        firstPassGenSet << <1, NUM_THREADS >> > (states, res, size, setsize, i * setsize);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }

    //Check for Duplicates
    /*
    for (int i = 0; i < split; i++) {
        secondPassGenSet << <1, NUM_THREADS >> > (states, res, size, setsize, i * setsize);
        printf("New Set %i\n", i);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
    }*/

    return res;

}

__host__ __device__
uint64_cu reformKey(addtype add, remtype rem, int N) {
    rem = rem << N;
    rem += add;
    return rem;
}

uint64_cu* generateCollidingSet(int size, int N) {
    
    #ifdef GPUCODE
        uint64_cu* res;
        gpuErrchk(hipMallocManaged(&res, size * sizeof(uint64_cu)));
    #else
        uint64_cu* res = new uint64_cu[size];
    #endif

    uint64_cu add = 7;

    for (int n = 0; n < (int) size/2; ++n) {
        uint64_cu num = reformKey(add, n, N);
        uint64_cu nval = RHASH_INVERSE(0, num);
        if (!contains(res, nval, n)) {
            res[n] = nval;
        }
        else {
            //Redo the step
            n--;
        }
    }

    add = 10;

    for (int n = ((int)size / 2); n < size; ++n) {
        uint64_cu num = reformKey(add, n, N);
        uint64_cu nval = RHASH_INVERSE(0, num);
        if (!contains(res, nval, n)) {
            res[n] = nval;
        }
        else {
            //Redo the step
            n--;
        }
    }

    return res;
}


//Sources: https://stackoverflow.com/questions/1894886/parsing-a-comma-delimited-stdstring
//         https://stackoverflow.com/questions/11876290/c-fastest-way-to-read-only-last-line-of-text-file
std::vector<std::string>* getLastArgs(std::string filename) {
    std::string line;
    std::ifstream infile;
    infile.open(filename);

    if (infile.is_open())
    {
        char ch;
        infile.seekg(-1, std::ios::end);        // move to location 65
        infile.get(ch);                         // get next char at loc 66
        if (ch == '\n')
        {
            infile.seekg(-2, std::ios::cur);    // move to loc 64 for get() to read loc 65
            infile.seekg(-1, std::ios::cur);    // move to loc 63 to avoid reading loc 65
            infile.get(ch);                     // get the char at loc 64 ('5')
            while (ch != '\n')                   // read each char backward till the next '\n'
            {
                infile.seekg(-2, std::ios::cur);
                infile.get(ch);
            }
            std::string lastLine;
            std::getline(infile, lastLine);
            std::cout << "The last line : " << lastLine << '\n';
            line = lastLine;
        }
        else
            printf("Exception:Check CSV format\n");
            throw std::exception();
    }
    else {
        printf("File failed to open\n");
        return nullptr;
    }

    std::vector<std::string>* vect = new  std::vector<std::string>;
    std::stringstream ss(line);
    std::string field;

    while (getline(ss, field, ',')) {
        vect->push_back(field);
    }

    for (std::size_t i = 0; i < vect->size(); i++){
        std::cout << vect->at(i) << std::endl;
    }

    return vect;
}

/*
 *
 * Main Functions
 *
 */

GPUHEADER_G
void fillClearyCuckoo(int N, uint64_cu* vals, ClearyCuckoo* H, addtype begin=0, int id=0, int s=1)
{
#ifdef GPUCODE
    int index = getThreadID();
    int stride = blockDim.x;
#else
    int index = id;
    int stride = s;
#endif
    
    for (int i = index+begin; i < N+begin; i += stride) {
        if (!(H->insert(vals[i]))) {
            break;
        }
    }
}

#ifdef GPUCODE
GPUHEADER_G
void fillClearyCuckoo(int N, uint64_cu* vals, ClearyCuckoo* H, addtype* occupancy, int* failFlag, int id = 0, int s = 1)
{
#ifdef GPUCODE
    int index = getThreadID();
    int stride = blockDim.x;
#else
    int index = id;
    int stride = s;
#endif

    for (int i = index; i < N; i += stride) {
        if (failFlag[0]) {
            break;
        }
        if (!(H->insert(vals[i]))) {
            atomicCAS(&(failFlag[0]), 0, 1);
            break;
        }
        atomicAdd(&occupancy[0], 1);
    }
}
#endif

GPUHEADER_G
void fillCleary(int N, uint64_cu* vals, Cleary* H, addtype begin=0, int id = 0, int s = 1)
{
#ifdef GPUCODE
    int index = getThreadID();
    int stride = blockDim.x;
#else
    int index = id;
    int stride = s;
#endif

    for (int i = index+begin; i < N+begin; i += stride) {
        if (!(H->insert(vals[i]))) {
            break;
        }
    }
}

GPUHEADER_G
void checkClearyCuckoo(int N, uint64_cu* vals, ClearyCuckoo* H, bool* res, int id = 0, int s = 1)
{
#ifdef GPUCODE
    int index = getThreadID();
    int stride = blockDim.x;
#else
    int index = id;
    int stride = s;
#endif

    for (int i = index; i < N; i += stride) {
        if (!(H->lookup(vals[i]))) {
            printf("\tSetting Res:Val %" PRIu64 " Missing\n", vals[i]);
            res[0] = false;
        }
    }
}

GPUHEADER_G
void checkCleary(int N, uint64_cu* vals, Cleary* H, bool* res, int id = 0, int s = 1)
{
#ifdef GPUCODE
    int index = getThreadID();
    int stride = blockDim.x;
#else
    int index = id;
    int stride = s;
#endif

    for (int i = index; i < N; i += stride) {
        if (!(H->lookup(vals[i]))) {
            printf("\tVal %" PRIu64 " Missing\n", vals[i]);
            res[0] = false;
        }
    }
}


bool TestFill(int N, int T, int tablesize, uint64_cu* vals, bool c, bool cc) {
    bool testres = true;

    //Init Var
    #ifdef GPUCODE
    bool* res;
    gpuErrchk(hipMallocManaged((void**)&res, sizeof(bool)));
    #else
    bool* res = new bool;
    #endif
    int numThreads = std::pow(2, T);

	//Create Table 1
    if (cc) {
#ifdef GPUCODE
        ClearyCuckoo* cc;
        gpuErrchk(hipMallocManaged((void**)&cc, sizeof(ClearyCuckoo)));
        new (cc) ClearyCuckoo(tablesize, 16);
#else
        ClearyCuckoo* cc = new ClearyCuckoo(tablesize, 16);
#endif

        printf("Filling ClearyCuckoo\n");
#ifdef GPUCODE
        fillClearyCuckoo << <1, 1 >> > (N, vals, cc);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
#else
        
        std::vector<std::thread> vecThread1(numThreads);

        for (int i = 0; i < numThreads; i++) {
            vecThread1.at(i) = std::thread(static_cast<void(*)(int, uint64_cu*, ClearyCuckoo*, addtype, int, int)>(fillClearyCuckoo), N, vals, cc, 0, i, numThreads);
        }

        //Join Threads
        for (int i = 0; i < numThreads; i++) {
            vecThread1.at(i).join();
        }
#endif
        printf("Devices Synced\n");
        cc->print();

        //Check Table
        res[0] = true;
        printf("Checking Cleary-Cuckoo\n");
#ifdef GPUCODE
        checkClearyCuckoo << <1, 1 >> > (N, vals, cc, res);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
#else
        checkClearyCuckoo(N, vals, cc, res);
#endif
        printf("Devices Synced\n");
        if (res[0]) {
            printf("All still in the table\n");
        }
        else {
            //testres = false;
            printf("!---------------------Vals Missing---------------------!\n");
        }
#ifdef GPUCODE
        gpuErrchk(hipFree(cc));
#else
        delete cc;
#endif
    }

    if (c) {
        //Create Table 2
#ifdef GPUCODE
        Cleary* c;
        gpuErrchk(hipMallocManaged((void**)&c, sizeof(Cleary)));
        new (c) Cleary(tablesize);
#else
        Cleary* c = new Cleary(tablesize);
#endif

        printf("Filling Cleary\n");
#ifdef GPUCODE
        fillCleary << <1, 1 > >> (N, vals, c);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
#else
        std::vector<std::thread> vecThread2(numThreads);

        for (int i = 0; i < numThreads; i++) {
            vecThread2.at(i) = std::thread(fillCleary, N, vals, c, 0, i, numThreads);
        }

        //Join Threads
        for (int i = 0; i < numThreads; i++) {
            vecThread2.at(i).join();
        }
#endif
        printf("Devices Synced\n");
        c->print();

        //Checking
        res[0] = true;
        printf("Checking Cleary\n");
#ifdef GPUCODE
        checkCleary << <1, 1 >> > (N, vals, c, res);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
#else
        checkCleary(N, vals, c, res);
#endif
        printf("Devices Synced\n");
        if (res[0]) {
            printf("All still in the table\n");
        }
        else {
            testres = false;
            printf("!---------------------Vals Missing---------------------!\n");
        }
#ifdef GPUCODE
        gpuErrchk(hipFree(c));
#else
        delete c;
#endif
    }

    //Destroy Vars
    #ifdef GPUCODE
        gpuErrchk(hipFree(res));
    #else
        delete res;
    #endif

        return testres;
}


GPUHEADER_G
void lockTestDevice(ClearyEntry<addtype, remtype>* T){
    addtype left = 1;
    addtype right = 4;

    while (true) {
        printf("\tGetting First Lock\n");
        if (!T[left].lock(false)) {
            printf("\tFirst Lock Failed\n");
                continue;
        }

        printf("\tLeft");
        T[left].print();

        printf("\tGetting Second Lock\n");
        if (!T[right].lock(false)) {
            printf("\tSecond Lock Failed\n");
                printf("\tAbort Locking\n");
            T[left].unlock();
            printf("\tUnlocked\n");
                continue;
        }

        printf("\tRight");
        T[left].print();

        printf("\t'Insertion\' Succeeded\n");
        T[left].unlock();
        T[right].unlock();
        printf("\tUnlocked\n");

        printf("\tLeft");
        T[left].print();
        printf("\tRight");
        T[left].print();

        return;
    }

}

void lockTest() {
    int tablesize = 256;
    ClearyEntry<addtype, remtype>* T;
    gpuErrchk(hipMallocManaged(&T, tablesize * sizeof(ClearyEntry<addtype, remtype>)));

    printf("\tInitializing Entries\n");
    for (int i = 0; i < tablesize; i++) {
        new (&T[i]) ClearyEntry<addtype, remtype>();
    }

    printf("\tStarting Lock Test\n");
#ifdef GPUCODE
    lockTestDevice << <1, 10 >> > (T);
#else

#endif
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk(hipFree(T));
}

void entryTest() {
    ClearyEntry<addtype, remtype> c{};
    c.setR(351629921636382);
    c.print();
    printf("Entry After R %" PRIu64 "\n", c.getR());
}

void Test(int N, int T, int L, bool c, bool cc) {
    bool res = true;

    const int addressSize = N;
    const int testSize = std::pow(2, addressSize);
    //const int testSize = 5;


    //printf("Lock Test\n");
    //lockTest();
    std::chrono::steady_clock::time_point begin;
    std::chrono::steady_clock::time_point end;


    begin = std::chrono::steady_clock::now();

    for (int i = 0; i < L; i++) {

        printf("==============================================================================================================\n");
        printf("                              BASIC TEST                              \n");
        printf("==============================================================================================================\n");
        uint64_cu* testset1 = generateTestSet(testSize);
        if (!TestFill(testSize, T, addressSize, testset1, c, cc)) {
            res = false;
        }
#ifdef GPUCODE
        gpuErrchk(hipFree(testset1));
#else
        delete[] testset1;
#endif

        printf("==============================================================================================================\n");
        printf("                            COLLISION TEST                            \n");
        printf("==============================================================================================================\n");
        uint64_cu* testset2 = generateCollidingSet(testSize, addressSize);
        if (!TestFill(testSize, T, addressSize, testset2, c, cc)) {
            res = false;
        }
#ifdef GPUCODE
        gpuErrchk(hipFree(testset2));
#else
        delete[] testset2;
#endif

        if (!res) {
            printf("TEST FAILED\n");
            break;
        }
        else {
            printf("TEST PASSED\n");
        }
    }

    end = std::chrono::steady_clock::now();

    if (res) {
        printf("==============================================================================================================\n");
        printf("                                             ALL TESTS PASSED                                                 \n");
        printf("==============================================================================================================\n");
    }
    std::cout << "Time Running:" << (std::chrono::duration_cast<std::chrono::seconds> (end - begin).count());
}


/* ================================================================================================================
 *
 *  Benchmark Methods
 *
 * ================================================================================================================
*/

void BenchmarkFilling(int NUM_TABLES_start, int NUM_TABLES, int INTERVAL, int NUM_SAMPLES, int NUM_THREADS, int NUM_LOOPS, int NUM_HASHES, std::vector<std::string>* params = nullptr) {

    const int WARMUP = 2;

    printf("=====================================================================\n");
    printf("                     Starting INSERTION BENCHMARK                    \n");
    printf("=====================================================================\n");

    std::ofstream myfile;
    std::string filename = "results/benchfill.csv";

    if (params) {
        printf("Opening\n");
        myfile.open(filename, std::ios_base::app);
        printf("Maybe\n");
    }
    else {
        myfile.open(filename);
    }

    if (!myfile.is_open()) {
        printf("File Failed to Open\n");

        return;
    }
    printf("File Opened\n");

    if (!params) {
        myfile << "tablesize,numthreads,loops,hashes,samples,type,interval,time\n";
    }

    printf("=====================================================================\n");
    printf("                     Starting Cleary-Cuckoo                \n\n");

    int NUM_GEN_THREADS = 256;

    //Tablesizes
    bool setup = true;
    for (int N = NUM_TABLES_start; N < NUM_TABLES_start + NUM_TABLES; N++) {
        if (params && setup) {
            N = std::stoi(params->at(0));
        }
        printf("Table Size:%i\n", N);

        int size = std::pow(2, N);
        int setsize = (int)(size / INTERVAL);

        if (setsize == 0) {
            printf("Error: Number of Intervals is greater than number of elements\n");
        }

        //Number of Threads
        for (int T = 0; T < NUM_THREADS; T++) {
            if (params && setup) {
                T = std::stoi(params->at(1));
            }
            printf("\tNumber of Threads:%i\n", T);
            //Number of Loops
            for (int L = 0; L < NUM_LOOPS; L++) {
                int numThreads = std::pow(2, T);

                if (params && setup) {
                    L = std::stoi(params->at(2));
                }
                printf("\t\tNumber of Loops:%i\n", L);
                //Number of Hashes
                for (int H = 1; H < NUM_HASHES; H++) {
                    printf("\t\t\tNumber of Hashes:%i\n", H);
                    if (params && setup) {
                        H = std::stoi(params->at(3));
                    }

                    //Number of samples
                    for (int S = 0; S < NUM_SAMPLES; S++) {
                        if (params && setup) {
                            S = std::stoi(params->at(4));
                        }
                        setup = false;
                        uint64_cu* vals = generateTestSetParallel(size, NUM_GEN_THREADS);
                        //Init Cleary Cuckoo

#ifdef GPUCODE
                        ClearyCuckoo* cc;
                        gpuErrchk(hipMallocManaged((void**)&cc, sizeof(ClearyCuckoo)));
                        new (cc) ClearyCuckoo(N, H);
#else
                        ClearyCuckoo* cc = new ClearyCuckoo(N, H);
#endif

                        cc->setMaxLoops(L);

                        //Loop over intervals
                        for (int j = 0; j < INTERVAL + WARMUP; j++) {
                            //Fill the table
                            std::chrono::steady_clock::time_point begin;
                            std::chrono::steady_clock::time_point end;

                            begin = std::chrono::steady_clock::now();
                            if (j >= WARMUP) {
#ifdef GPUCODE                  
                                fillClearyCuckoo << <1, std::pow(2, T) >> > (setsize, vals, cc, setsize * (j - WARMUP));
                                gpuErrchk(hipPeekAtLastError());
                                gpuErrchk(hipDeviceSynchronize());
#else
                                int numThreads = T+1;
                                std::vector<std::thread> vecThread(numThreads);
                                for (int i = 0; i < numThreads; i++) {
                                    vecThread.at(i) = std::thread(static_cast<void(*)(int, uint64_cu*, ClearyCuckoo*, addtype, int, int)>(fillClearyCuckoo), setsize, vals, cc, setsize * (j - WARMUP), i, numThreads);
                                }

                                //Join Threads
                                for (int i = 0; i < numThreads; i++) {
                                    vecThread.at(i).join();
                                }
#endif
                                //End the timer
                                end = std::chrono::steady_clock::now();

                                myfile << N << "," << numThreads << "," << L << "," << H << "," << S << ",cuc," << (j - WARMUP) << "," << (std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count()) / setsize << ",\n";
                            }

                        }
#ifdef GPUCODE
                        gpuErrchk(hipFree(cc));
                        gpuErrchk(hipFree(vals));
#else       
                        delete cc;
                        gpuErrchk(hipFree(vals));
#endif
                    }
                }
            }
        }
    }

    printf("=====================================================================\n");
    printf("                     Starting Cleary                \n\n");

    for (int N = NUM_TABLES_start; N < NUM_TABLES_start + NUM_TABLES; N++) {
        if (params && setup) {
            N = std::stoi(params->at(0));
        }
        printf("Table Size:%i\n", N);

        int size = std::pow(2, N);
        int setsize = (int)(size / INTERVAL);
        for (int T = 0; T < NUM_THREADS; T++) {
            printf("\tNumber of Threads:%i\n", T);
            for (int S = 0; S < NUM_SAMPLES; S++) {
                printf("\t\t\t\tSample Number:%i\n", S);
                uint64_cu* vals = generateTestSetParallel(size, NUM_GEN_THREADS);

                //Init Cleary
                #ifdef GPUCODE
                Cleary* c;
                gpuErrchk(hipMallocManaged((void**)&c, sizeof(Cleary)));
                new (c) Cleary(N);
                #else
                Cleary* c = new Cleary(N);
                #endif

                //Loop over intervals
                for (int j = 0; j < INTERVAL + WARMUP; j++) {
                  std::chrono::steady_clock::time_point begin;
                  std::chrono::steady_clock::time_point end;

                    //Fill the table
                    begin = std::chrono::steady_clock::now();
                    if (j >= WARMUP) {
                        
                        #ifdef GPUCODE
                            int numThreads = std::pow(2, T);
                            fillCleary << <1, numThreads >> > (setsize, vals, c, setsize* (j - WARMUP));
                            gpuErrchk(hipPeekAtLastError());
                            gpuErrchk(hipDeviceSynchronize());
                        #else
                            int numThreads = T+1;
                            std::vector<std::thread> vecThread(numThreads);
                            
                            for (int i = 0; i < numThreads; i++) {
                                vecThread.at(i) = std::thread(fillCleary, setsize, vals, c, setsize * (j - WARMUP), i, numThreads);
                            }

                            //Join Threads
                            for (int i = 0; i < numThreads; i++) {
                                vecThread.at(i).join();
                            }
                        #endif
                        //End the timer
                        end = std::chrono::steady_clock::now();
                        myfile << N << "," << numThreads << "," << -1 << "," << -1 << "," << S << ",cle," << (j - WARMUP) << "," << (std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count()) / setsize << ",\n";
                    }

                }
                #ifdef GPUCODE
                gpuErrchk(hipFree(c));
                gpuErrchk(hipFree(vals));
                #else       
                delete c;
                gpuErrchk(hipFree(vals));
                #endif
            }
        }
    }

    myfile.close();
    printf("\nBenchmark Done\n");
}

void BenchmarkMaxOccupancy(int TABLESIZES, int NUM_HASHES, int NUM_LOOPS, int NUM_SAMPLES) {

    printf("=====================================================================\n");
    printf("                   Starting MAX Occupancy Benchmark                  \n");
    printf("=====================================================================\n");

    std::ofstream myfile;
    std::string filename = "results/benchmax.csv";
    myfile.open(filename);
    if (!myfile.is_open()) {
        printf("File Failed to Open\n");
        return;
    }
    printf("File Opened");

    myfile << "tablesize,numhashes,numloops,samples,max\n";

    //MAX_LOOPS
    for (int N = 5; N < 5 + TABLESIZES; N++) {
        printf("Table Size:%i\n", N);
        int size = std::pow(2, N);
        for (int j = 1; j < NUM_HASHES; j++) {
            printf("\tNum of Hashes:%i\n", j);
            for (int k = 0; k < NUM_LOOPS; k++) {
                printf("\t\tNum of Loops:%i\n", k);
                for (int S = 0; S < NUM_SAMPLES; S++) {
                    //printf("\t\t'tSample Number:%i\n", S);
                    uint64_cu* vals = generateTestSet(size);

                    int* failFlag;
                    gpuErrchk(hipMallocManaged(&failFlag, sizeof(int)));
                    failFlag[0] = false;

                    //Init Cleary Cuckoo
                    ClearyCuckoo* cc;
                    gpuErrchk(hipMallocManaged((void**)&cc, sizeof(ClearyCuckoo)));
                    new (cc) ClearyCuckoo(N, j);
                    cc->setMaxLoops(k);

                    //Var to store num of inserted values
                    addtype* occ;
                    gpuErrchk(hipMallocManaged(&occ, sizeof(addtype)));
                    occ[0] = 0;

                    //Fill the table
#ifdef GPUCODE
                    fillClearyCuckoo << <1, 256 >> > (size, vals, cc, occ, failFlag);
                    gpuErrchk( hipPeekAtLastError() );
                    gpuErrchk( hipDeviceSynchronize() );

                    myfile << N << "," << j << "," << k << "," << S << "," << occ[0] << ",\n";

                    gpuErrchk(hipFree(failFlag));
                    gpuErrchk(hipFree(cc));
                    gpuErrchk(hipFree(occ));
                    gpuErrchk(hipFree(vals));
#else
                    delete failFlag;
                    delete cc;
                    delete occ;
                    delete[] vals;
#endif
                }
            }
        }
    }

    myfile.close();

    printf("\n\nBenchmark Done\n");
}


int main(int argc, char* argv[])
{
    if (argc == 1) {
        printf("No Arguments Passed\n");
    }

    if (strcmp(argv[1], "test") == 0) {
        bool c = false;
        bool cc = false;

        if (argc < 5) {
            printf("Not Enough Arguments Passed\n");
            printf("Required: TABLESIZE, NUM_THREADS, SAMPLES, TABlETYPE (c cc ccc)\n");
            return 0;
        }

        std::string s = argv[5];
        c = s == "c";
        cc = s == "cc";
        if (s == "ccc") {
            c = true;
            cc = true;
        }

        Test(std::stoi(argv[2]), std::stoi(argv[3]), std::stoi(argv[4]), c, cc);
    }
    else if (strcmp(argv[1], "benchmax") == 0) {
        if (argc < 6) {
            printf("Not Enough Arguments Passed\n");
            printf("Required: TABLESIZES, NUM_HASHES, NUM_LOOPS, NUM_SAMPLES\n");
            return 0;
        }
        BenchmarkMaxOccupancy(std::stoi(argv[2]), std::stoi(argv[3]), std::stoi(argv[4]), std::stoi(argv[5]));
    }
    else if (strcmp(argv[1], "benchfill") == 0) {
        if (argc < 7) {
            printf("Not Enough Arguments Passed\n");
            printf("Required: NUM_TABLES start, end, INTERVAL, NUM_SAMPLES, NUM_THREADS, NUM_LOOPS, NUM_HASHES\n");
            return 0;
        }
        else if (strcmp(argv[2], "continue") == 0) {
            printf("Continuing from Last Position\n");
            std::vector<std::string>* lastargs = getLastArgs("results/benchfill.csv");

            BenchmarkFilling(std::stoi(argv[3]), std::stoi(argv[4]), std::stoi(argv[5]), std::stoi(argv[6]), std::stoi(argv[7]), std::stoi(argv[8]), std::stoi(argv[9]), lastargs);
            delete lastargs;
            return 0;
        }

        BenchmarkFilling(std::stoi(argv[2]), std::stoi(argv[3]), std::stoi(argv[4]), std::stoi(argv[5]), std::stoi(argv[6]), std::stoi(argv[7]), std::stoi(argv[8]));
    }

    else if (strcmp(argv[1], "debug") == 0) {
        int NUM_THREADS = 8;

        uint64_cu* test2 = generateTestSetParallel(10000, NUM_THREADS);
        printf("Generated:\n");
        for (int i = 0; i < 10000; i++) {
            printf("%i: %" PRIu64 "\n",i, test2[i]);
        }
    }

    return 0;
}
