#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <stdlib.h>
#include <math.h>
#include <string>
#include <math.h>
#include <assert.h>

#include <bitset>
#include <inttypes.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "int_cu.h"

#ifndef HASHTABLE
#define HASHTABLE
#include "HashTable.h"
#endif

#ifndef HASHINCLUDED
#define HASHINCLUDED
#include "hashfunctions.cu"
#endif

#include "ClearyEntry.cu"

//Types to allow for changes

using addtype = uint32_t;
using remtype = uint64_cu;
using hashtype = uint64_cu;
using keytype = uint64_cu;

//Enum for searching

enum direction{up, down, here};


class Cleary : public HashTable{
    //Allows for easy changing of the types

    private:
        //Constant Vars
        const static int HS = 59;       //HashSize
        const static int BUFFER = 0; //Space assigned for overflow
        const static int MAXLOOPS = 24;
        const static int A_UNDEFINED = 0;

        bool GPU;

        //Vars assigned at construction
        int AS;                  //AdressSize
        int RS;                  //RemainderSize
        int size;                //Allocated Size of Table
        int tablesize;              //Actual size of table with buffer
        int occupancy = 0;

        addtype MAX_ADRESS;
        addtype MIN_ADRESS = 0;

        //Tables
        ClearyEntry<addtype, remtype>* T;

        //Hash function ID
        int h1;

        GPUHEADER
            addtype getAdd(keytype key) {
            hashtype mask = ((hashtype)1 << AS) - 1;
            addtype add = key & mask;
            return add;
        }

        GPUHEADER
            remtype getRem(keytype key) {
            remtype rem = key >> AS;
            return rem;
        }

        GPUHEADER
            uint64_cu reformKey(addtype add, remtype rem) {
            rem = rem << AS;
            rem += add;
            return rem;
        }

        GPUHEADER
        addtype findIndex(uint64_cu k){           
            hashtype h = RHASH(h1, k);
            addtype j = getAdd(h);
            remtype rem = getRem(h);

            addtype i = j;
            int cnt = 0;

            //printf("\t\t\t\t\t\t\t\t\t\t\t%i: Finding Index from %" PRIu32 "\n", getThreadID(), i);

            //Find first well defined A value
            while(T[i].getA() == A_UNDEFINED && i>=MIN_ADRESS && T[i].getO()){
                cnt = cnt - (T[i].getV() ? 1 : 0);
                i=i-1;
                if (i > MAX_ADRESS) {
                    break;
                }
            };

            //printf("\t\t\t\t\t\t\t\t\t\t\t%i: First well defined: %" PRIu32 "\n", getThreadID(), i);
            if (i <= MAX_ADRESS && i >= MIN_ADRESS) {
                cnt = cnt + T[i].getA();
                //printf("\t\t\t\t\t\t\t\t\t\t\t%i: Cnt: %i\n", getThreadID(), cnt);
            }

            //Look for the relevant group
            //printf("\t\t\t\t\t\t\t\t\t\t\t%i: Find relevant group\n", getThreadID());
            direction dir = up;
            if(cnt < 0){
                dir = up;
                //printf("\t\t\t\t\t\t\t\t\t\t\t%i: Dir Up %" PRIu32 "\n", getThreadID(), i);
                while(cnt != 0 && i != MAX_ADRESS){
                    i = i+1;
                    cnt = cnt + (T[i].getC() ? 1 : 0);
                };
                if(T[i].getR() >= rem){
                    dir = here;
                }
            }else if(cnt > 0){
                dir = down;
                //printf("\t\t\t\t\t\t\t\t\t\t\t%i: Dir down %" PRIu32 "\n", getThreadID(), i);
                while(cnt != 0 && i != MIN_ADRESS){
                    cnt = cnt - (T[i].getC() ? 1 : 0);
                    i = i - 1;
                }
                if(T[i].getR() <= rem){dir = here;}
            }else{
                //printf("\t\t\t\t\t\t\t\t\t\t\t%i: End Case\n", getThreadID());
                if (i > MAX_ADRESS) {
                    i = 0;
                    //IF val is being inserted first time, stop here
                    if (!T[j].getV()) {
                        return i;
                    }
                }

                if(T[i].getR() > rem){dir = down;}
                else if(T[i].getR() < rem){dir = up;}
                else{
                    //printf("\t\t\t\t\t\t\t\t\t\t\t%i: End Else Case\n", getThreadID());
                    dir = here;}
            }
            //printf("\t\t\t\t\t\t\t\t\t\t\t%i: Relevant Group: %" PRIu32 "\n", getThreadID(), i);

            //Look inside of the group
            //printf("\t\t\t\t\t\t\t\t\t\t\t%i: Look inside group\n", getThreadID());
            switch (dir){
                case here:
                    //printf("\t\t\t\t\t\t\t\t\t\t\t%i: Here\n", getThreadID());
                    break;

                case down:
                    while (dir != here) {
                        assert(i <= MAX_ADRESS);
                        //printf("\t\t\t\t\t\t\t\t\t\t\t%i: Going Down %" PRIu32 "\n", getThreadID(), i);
                        if (T[i].getC() == 1 || i == MIN_ADRESS) { dir = here; }
                        else {
                            i = i - 1;
                            if (T[i].getR() <= rem) {
                                dir = here;
                            }
                        }
                    }
                    break;

                case up:
                    while (dir != here) {
                        assert(i <= MAX_ADRESS);
                        //printf("\t\t\t\t\t\t\t\t\t\t\t%i: Going Up %" PRIu32 "\n", getThreadID(), i);
                        if (i == MAX_ADRESS) {
                            dir = here;
                        }
                        else if (T[i + 1].getC() == 1) {
                            dir = here;
                        }
                        else {
                            i = i + 1;
                            if (T[i].getR() >= rem) {
                                dir = here;
                            }
                        }
                    }
                    break;

                default:
                    break;
            };
            return i;
        }

        GPUHEADER
        addtype leftLock(addtype i) {
            if (i == MIN_ADRESS) {
                return i;
            }
            while (T[i].getO() && i>MIN_ADRESS) {
                i -= 1;
            }
            return i;
        }

        GPUHEADER
        addtype rightLock(addtype i) {
            if (i == MAX_ADRESS) {
                return i;
            }
            while (T[i].getO() && i<MAX_ADRESS) {
                i += 1;
            }
            return i;
        }


        GPUHEADER
        bool insertIntoTable(keytype k, addtype left, addtype right) {
            //printf("\t\t\t\t\t\t\t%i: Inserting Into Table\n", getThreadID());

            hashtype h = RHASH(h1, k);
            addtype j = getAdd(h);
            remtype rem = getRem(h);

            bool newgroup = false;

            //Find insertion index
            addtype i = findIndex(k);
            //printf("\t\t\t\t\t\t\t%i: Index Found %" PRIu32 "\n", getThreadID(), i);

            //Check virgin bit and set
            if (!T[j].getV()) {
                //printf("\t\t\t\t\t\t\t%i: Set VBit at %" PRIu32 "\n", getThreadID(), j);
                T[j].setV(true);
                newgroup = true;
            }

            //printf("\t\t\t\t\t\t\t%i: Group Start/Groupend\n", getThreadID());
            bool groupstart = T[i].getC() == 1 && T[i].getO() != false;
            bool groupend;
            if (i != MAX_ADRESS) { groupend = T[i + 1].getC() == 1 && T[i].getO() != false; }
            else { groupend = true; }

            //Check whether i should be 0 (Check all smaller Vs
            //printf("\t\t\t\t\t\t\t\t\t\t%i: Check if i is 0 \n", getThreadID());
            bool setStart = false;
            
            if (i == MIN_ADRESS && j != MIN_ADRESS && !T[MIN_ADRESS].getV()) {
                setStart = true;
                for (int x = 1; x < j; x++) {
                    if (T[x].getV() != 0) {
                        setStart = false;
                        break;
                    }
                }
            }
            //printf("\t\t\t\t\t\t\t\t\t\t%i: Look for new group\n",getThreadID());
            //If a new group needs to be formed, look for the end of the group
            if (newgroup && T[i].getO() && !setStart) {
                direction dir = up;
                while (dir != here) {
                    if (i == MAX_ADRESS) {
                        dir = here;
                    }
                    else if (T[i + 1].getC() == 1) {
                        i++;
                        dir = here;
                    }
                    else {
                        i = i + 1;
                    }
                };
            }

            //Decide to shift mem up or down
            //TODO: Maybe randomize
            int shift = 1;
            //printf("\t\t\t\t\t\t\t\t\t\t%i: Prevent Overflows %" PRIu32 "\n", getThreadID(), i);
            //Prevent Overflows
            if (T[left].getO()) {
                shift = 1;
            }
            else if (T[right].getO()) {
                shift = -1;
            }
            

            //Edge cases where the location must be shifted
            //printf("\t\t\t\t\t\t\t\t\t\t%i: Edge Cases %" PRIu32 "\n", getThreadID(), i);
            bool setC = false;
            if (shift == -1) {
                //printf("\t\t\t\t\t\t\t\t\t\t%i: Shift -1\n",getThreadID());
                if (groupstart && (!newgroup) && (T[i].getR() > rem) && T[i].getO() && (i != MIN_ADRESS)) {
                    //printf("\t\t\t\t\t\t\t\t\t\t%i: Shift Case 1\n",getThreadID());
                    T[i].setC(false);
                    setC = true;
                    i--;
                }
                else if (!newgroup && T[i].getR() > rem && T[i].getO() && i != MIN_ADRESS) {
                    //printf("\t\t\t\t\t\t\t\t\t\t%i: Shift Case 2\n",getThreadID());
                    i--;
                }
                else if (newgroup && T[i].getO() && i != MIN_ADRESS) {
                    //printf("\t\t\t\t\t\t\t\t\t\t%i: Shift Case 3\n",getThreadID());
                    if (i == MAX_ADRESS && j != MAX_ADRESS) {
                        bool checkPos = true;
                        for (int m = j + 1; m <= MAX_ADRESS; m++) {
                            if (T[m].getV()) { checkPos = false; break; }
                        }
                        if (!checkPos) {
                            i--;
                        }
                    }
                    else if (i != MAX_ADRESS) {
                        //printf("\t\t\t\t\t\t\t\t\t\t%i: Shift Case 4\n",getThreadID());
                        i--;
                    }
                }
            }
            if (shift == 1) {
                //printf("\t\t\t\t\t\t\t\t\t\t%i: Shift 1\n", getThreadID());
                if (groupend && (!newgroup) && (T[i].getR() < rem) && T[i].getO() && (i != MAX_ADRESS)) {
                    //printf("\t\t\t\t\t\t\t\t\t\t%i: Shift Case 5:%" PRIu32 "\n",getThreadID(),i);
                    i++;
                    //printf("\t\t\t\t\t\t\t\t\t\t%i: Iter Case 5:%" PRIu32 "\n",getThreadID(),i);
                    T[i].setC(false);
                    //printf("\t\t\t\t\t\t\t\t\t\t%i: SetC 5:%" PRIu32 "\n",getThreadID(),i);
                    setC = true;
                    //printf("\t\t\t\t\t\t\t\t\t\t%i: Done Shift Case 5\n",getThreadID());
                }
                else if (!newgroup && T[i].getR() < rem && T[i].getO() && i != MAX_ADRESS) {
                    //printf("\t\t\t\t\t\t\t\t\t\t%i: Shift Case 6\n",getThreadID());
                    i++;
                }
                else if (j == MIN_ADRESS && newgroup) {
                    //printf("\t\t\t\t\t\t\t\t\t\t%i: Shift Case 7\n",getThreadID());
                    i = MIN_ADRESS;
                }
            }

            //printf("\t\t\t\t\t\t\t\t\t\t%i: Storing searchstart %" PRIu32 "\n", getThreadID(), i);

            //Store where the search started for later
            addtype startloc = i;
            assert(0<=i && i<=MAX_ADRESS);

            //Check whether location is empty
            bool wasoccupied = T[i].getO();

            //printf("\t\t\t\t\t\t\t\t\t\t%i: Storing old Values at %" PRIu32 "\n", getThreadID(), i);
            //Store values at found location
            remtype R_old = T[i].getR();
            bool C_old = T[i].getC();
            bool O_old = T[i].getO();

            //Insert new values
            //printf("\t\t\t\t\t\t\t\t\t\t%i: Setting new Values at %" PRIu32 "\n", getThreadID(), i);
            T[i].setR(rem);

            T[i].setO(true);
            if ((shift == 1) && !setC) {
                T[i].setC(C_old);
            }
            else if (shift == -1) {
                T[i].setC(newgroup);
            }

            //printf("\t\t\t\t\t\t\t\t\t\t%i: Update C %" PRIu32 "\n", getThreadID(), i);
            if (setC && shift == -1) { T[i].setC(true); }
            //Update C Value
            if (shift == 1 && !newgroup) {
                C_old = setC;
            }

            //If the space was occupied shift mem
            //printf("\t\t\t\t\t\t\t\t\t\t%i: Shifting Mem from %" PRIu32 "\n", getThreadID(), i);
            if (wasoccupied) {
                while (O_old) {
                    //printf("\t\t\t\t\t\t\t\t\t\t%i: Shift%" PRIu32 "\n", getThreadID(), i);
                    i += shift;
                    assert(0<=i && i<=MAX_ADRESS);
                    //Store the values
                    remtype R_temp = T[i].getR();
                    bool C_temp = T[i].getC();
                    bool O_temp = T[i].getO();

                    //Put the old values in the new location
                    T[i].setR(R_old);
                    T[i].setO(true);
                    T[i].setC(C_old);

                    //Store the old values again
                    R_old = R_temp;
                    C_old = C_temp;
                    O_old = O_temp;

                    if (i == MIN_ADRESS || i == MAX_ADRESS) {
                        break;
                    }

                }
            }

            if(A_UNDEFINED != 0){
                //Find the first well-defined A
                //printf("\t\t\t\t\t\t\t\t\t\t%i: Find Start of Group %" PRIu32 "\n", getThreadID(), i);
                addtype x = startloc;
                while(T[x].getA() == A_UNDEFINED && x!=MIN_ADRESS) {
                    x--;
                }
                if (x != MAX_ADRESS && !T[x].getO()) {
                    x++;
                }

                //Update the A values
                //printf("\t\t\t\t\t\t\t\t\t\t%i: Updating A from %" PRIu32 "\n", getThreadID(), x);
                int A_old = 0;
                while (T[x].getO() && x <= MAX_ADRESS) {
                    //printf("\t\t\t\t\t\t\t\t\t\t\t%i: Setting A %" PRIu32 "\n", getThreadID(), x);
                    assert(0<=x && x<=MAX_ADRESS);
                    //Update Based on C and V
                    if (T[x].getC()) {
                        A_old += 1;
                    }
                    if (T[x].getV()) {
                        A_old -= 1;
                    }
                    T[x].setA(A_old);
                    x++;
                    if (x > MAX_ADRESS) {
                        break;
                    }
                }
            }

            //printf("\t\tAfterallupdates");

            return true;
        }


    public:
        /**
         * Constructor
         */

        //Default constructor for mem-alloc
        Cleary() {}

        Cleary(int adressSize){
            //printf("Creating Cleary Table\n");
            AS = adressSize;
            RS = HS-AS;
            tablesize = (int) pow(2,AS) + 2*BUFFER;
            size = (int) pow(2,AS);
            MAX_ADRESS = tablesize - 1;

            //printf("\tAllocating Memory\n");
            #ifdef GPUCODE
            gpuErrchk(hipMallocManaged(&T, tablesize * sizeof(ClearyEntry<addtype, remtype>)));
            #else
            T = new ClearyEntry<addtype, remtype>[tablesize];
            #endif

            //printf("\tInitializing Entries\n");
            for(int i=0; i<tablesize; i++){
                new (&T[i]) ClearyEntry<addtype, remtype>();
            }

            h1 = 0;
            //printf("\tDone\n");
        }

        /**
         * Destructor
         */
        ~Cleary() {
            #ifdef GPUCODECODE
            gpuErrchk(hipFree(T));
            #else
            delete[] T;
            #endif
        }

        GPUHEADER_D
        bool insert(keytype k){
            //printf("\tInserting %" PRIu64 "\n", k);
            //Calculate Hash
            hashtype h = RHASH(h1, k);
            addtype j = getAdd(h);
            remtype rem = getRem(h);

            int counter=0;

            while (true) {
                //__syncthreads();
                counter++;
                assert(0<=j && j<=MAX_ADRESS);
                //assert(counter < 30000);

                //Try Non-Exclusive Write
                //printf("\t\t\t\t\t\t\t\t%i: Trying Non-Exclusive Write at %" PRIu32 "\n", getThreadID(), j);
                ClearyEntry<addtype, remtype> def(0, false, false, true, 0, false, false);
                ClearyEntry<addtype, remtype> newval(rem, true, true, true, 0, false, false);

                ClearyEntry<addtype, remtype> old(T[j].compareAndSwap(&def, &newval));

                //If not locked + not occupied then success
                if ((!old.getL()) && (!old.getO())) {
                    //printf("\t\t\t\t\t\t\t\t%i: Non-Exclusive Success\n", getThreadID());
                    return true;
                }

                //Else Need Exclusivity
                addtype left = leftLock(j);
                addtype right = rightLock(j);

                assert(0<=left && left<=MAX_ADRESS);
                assert(0<=right && right<=MAX_ADRESS);

                if (!T[left].lock(left == MIN_ADRESS)) {
                    //printf("\t\t\t\t\t\t\t\t%i: Left Failed at%" PRIu32 "\n", getThreadID(), left);
                    //__nanosleep(1000);
                    continue;
                }
                //printf("\t\t\t\t\t\t\t\t%i: Left Retrieved at%" PRIu32 "\n", getThreadID(), left);
                //printf("\t\t\t\t\t\t\t\t%i: Trying Right at%" PRIu32 "\n", getThreadID(), right);
                if (!T[right].lock(right == MAX_ADRESS)) {
                    //printf("\t\t\t\t\t\t\t\t%i: Right Failed at%" PRIu32 "\n", getThreadID(), right);
                    T[left].unlock();
                    //printf("\t\t\t\t\t\t\t\t%i: Left Unlocked at%" PRIu32 "\n", getThreadID(), left);
                    //__nanosleep(1000);
                    continue;
                }
                //printf("\t\t\t\t\t\t\t\t%i: Right Retrieved at%" PRIu32 "\n", getThreadID(), right);

                //Do a read
                if (lookup(k)) {
                    //Val already exists
                    //printf("\t\tVal Already Exists\n");
                    T[left].unlock();
                    T[right].unlock();
                    //printf("\t\t\t\t\t\t\t\t%i: Left Unlocked at%" PRIu32 "\n", getThreadID(), left);
                    //printf("\t\t\t\t\t\t\t\t%i: Right Unlocked at%" PRIu32 "\n", getThreadID(), right);
                    return false;
                }

                //Write
                //printf("\t\t\t\t\t\t\t\t%i: Exclusive Write\n", getThreadID());
                bool res = insertIntoTable(k, left, right);
                T[left].unlock();
                T[right].unlock();
                //printf("\t\t\t\t\t\t\t\t%i: Left Unlocked at%" PRIu32 "\n", getThreadID(), left);
                //printf("\t\t\t\t\t\t\t\t%i: Right Unlocked at%" PRIu32 "\n", getThreadID(), right);
                //printf("\t\t\t\t\t\t\t\t%i: Insertion Success\n", getThreadID());
                //printf("\tAfterInsertion");
                return res;
            }
        };

        GPUHEADER
        bool lookup(uint64_cu k){
            //printf("\t\tLookup %" PRIu64 "\n", k);
            //Hash Key
            hashtype h = RHASH(h1, k);
            addtype j = getAdd(h);
            remtype rem = getRem(h);

            //If no values with add exist, return
            if(T[j].getV() == 0){
                //printf("\t\t\tV not set\n");
                return false;
            };

            addtype i = findIndex(k);
            assert(0<=i && i<=MAX_ADRESS);
            //printf("\t\tFind Add   %" PRIu32 "\n", j);
            //printf("\t\tFind Index %" PRIu32 "\n", i);

            if(T[i].getR() == rem){
                return true;
            }
            else {
                //printf("\t\t\tOriginalIndex:%" PRIu32 " FoundIndex:%" PRIu32 "\n",j, i);
                //printf("\t\t\tFoundR:%" PRIu64 " ActualR:%" PRIu64 "\n", T[i].getR(), rem);
            }

            return false;
        };

        GPUHEADER
        void clear(){
            for(int i=0; i<tablesize; i++){
                new (&T[i]) ClearyEntry<addtype, remtype>();
            }
        }

        GPUHEADER
        int getSize(){
            return size;
        }

        GPUHEADER
        void print(){
            printf("----------------------------------------------------------------\n");
            printf("|    i     |     R[i]       | C[i] | V[i] | O[i] | A[i] | L[i] |\n");
            printf("----------------------------------------------------------------\n");
            for(int i=0; i<tablesize; i++){
                if(true){
                    printf("|%-10i|%-16" PRIu64 "|%-6i|%-6i|%-6i|%-6i|%-6i|\n", i, T[i].getR(), T[i].getC(), T[i].getV(), T[i].getO(), T[i].getA(), T[i].getL());
                }
            }
            printf("----------------------------------------------------------------\n");
        }

        //No rehash
        GPUHEADER
        bool rehash(){return true;}

};
