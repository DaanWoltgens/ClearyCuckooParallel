#include "hip/hip_runtime.h"
#include "int_cu.h"


#ifndef ENTRYINCLUDED
#define ENTRYINCLUDED
#include "TableEntry.h"
#endif
#include <math.h>


template <class ADD, class REM>
class ClearyEntry : TableEntry <ADD, REM> {

private:
    int Rindex[2] = {  1, 56 };
    int Oindex[2] = { 57, 57 };
    int Vindex[2] = { 58, 58 };
    int Cindex[2] = { 59, 59 };
    int Lindex[2] = { 60, 60 };
    int Aindex[2] = { -1, -1 };


public:
    __host__ __device__
    ClearyEntry(REM R, bool O, bool V, bool C, int A, bool L, bool onDevice = true) {
        TableEntry<ADD, REM>::val = 0;
        setR(R, onDevice);
        setO(O, onDevice);
        setV(V, onDevice);
        setC(C, onDevice);
        setA(A, onDevice);
        setL(L, onDevice);
        return;
    }

    __host__ __device__
    ClearyEntry(uint64_cu x) {
        TableEntry<ADD, REM>::val = x;
        return;
    }

    __host__ __device__
    ClearyEntry() : ClearyEntry(0, false, false, true, 0, false) {}

    __host__ __device__
    void exchValue(ClearyEntry* x) {
        //Atomically set this value to the new one
        uint64_cu old = atomicExch(TableEntry<ADD, REM>::getValPtr(), x->getValue());
        //Return an entry with prev val
        x->setValue(old);
        return;
    }

    __host__ __device__
    void setR(REM x, bool onDevice = true) {
        TableEntry<ADD, REM>::setBits(Rindex[0], Rindex[1], x, onDevice);
        return;
    }

    __host__ __device__
    REM getR() {
        return (REM)TableEntry<ADD, REM>::getBits(Rindex[0], Rindex[1]);
    }

    __host__ __device__
    void setO(bool x, bool onDevice = true) {
        TableEntry<ADD, REM>::setBits(Oindex[0], Oindex[1], x, onDevice);
        return;
    }

    __host__ __device__
    bool getO() {
        return (bool)TableEntry<ADD, REM>::getBits(Oindex[0], Oindex[1]);
    }

    __host__ __device__
    void setV(bool x, bool onDevice = true) {
        TableEntry<ADD, REM>::setBits(Vindex[0], Vindex[1], x, onDevice);
        printf("\t\t\t\t\t\t\t\t\t\tV is Set\n");
        return;
    }

    __host__ __device__
    bool getV() {
        return (bool)TableEntry<ADD, REM>::getBits(Vindex[0], Vindex[1]);
    }

    __host__ __device__
    void setC(bool x, bool onDevice = true) {
        TableEntry<ADD, REM>::setBits(Cindex[0], Cindex[1], x, onDevice);
        return;
    }

    __host__ __device__
    bool getC() {
        return (bool)TableEntry<ADD, REM>::getBits(Cindex[0], Cindex[1]);
    }

    __host__ __device__
    void setA(int x, bool onDevice = true) {
        int Amin = -pow(2, (Aindex[1] - Aindex[0]) - 1);
        int Amax = pow(2, (Aindex[1] - Aindex[0]) - 1);

        //printf("Amin:%i Amax:%i", Amin, Amax);

        if (x > Amax-1) {
            x = Amax;
        }
        if (x < Amin) {
            x = Amax;
        }

        TableEntry<ADD, REM>::setBits(Aindex[0], Aindex[1], TableEntry<ADD, REM>::signed_to_unsigned(x, Aindex[1]-Aindex[0]), onDevice);

        return;
    }

    __host__ __device__
    int getA() {
        return TableEntry<ADD, REM>::unsigned_to_signed(TableEntry<ADD, REM>::getBits(Aindex[0], Aindex[1]), Aindex[1] - Aindex[0]);
    }

    __host__ __device__
    void setL(bool x, bool onDevice = true) {
        TableEntry<ADD, REM>::setBits(Lindex[0], Lindex[1], x, onDevice);
        return;
    }

    __host__ __device__
    bool getL() {
        return TableEntry<ADD, REM>::getBits(Lindex[0], Lindex[1]);
    }

    //Need to do with CAS
    __host__ __device__
    bool lock() {
        //Store old TableEntry<ADD, REM>::value
        uint64_cu oldval = TableEntry<ADD, REM>::val;
        printf("\t\t\t\t\t\t\t\t\t%i: Lock-Creating new Val\n", threadIdx.x);
        //Make the new value with lock locked
        uint64_cu newval = TableEntry<ADD, REM>::val;
        TableEntry<ADD, REM>::setBits(Lindex[0], Lindex[1], ((uint64_cu) 1), &newval, false);

        //If Lockbit was set return false
        if (TableEntry<ADD, REM>::getBits(Lindex[0], Lindex[1], oldval)) {
            //printf("\t\t\tLockbit Already Set\n");
            return false;
        }
        printf("\t\t\t\t\t\t\t\t\t%i: Lock-Swapping\n", threadIdx.x);
        //Swap if the old value hasn't changed
        uint64_cu res = atomicCAS(TableEntry<ADD, REM>::getValPtr(), oldval, newval);

        if(res == oldval){
          printf("\t\t\t\t\t\t\t\t\t%i: Lock-Success\n", threadIdx.x);
          return true;
        }
        else {
            printf("\t\t\t\t\t\t\t\t\t%i: Lock-Fail\n", threadIdx.x);
            return false;
        }
    }

    __host__ __device__
    bool unlock() {
        //Swap if the old value hasn't changed
        while(true){
          //Store old Value
          uint64_cu oldval = TableEntry<ADD, REM>::val;
          //Make the new value with lock unlocked
          uint64_cu newval = TableEntry<ADD, REM>::val;
          TableEntry<ADD, REM>::setBits(Lindex[0], Lindex[1], ((uint64_cu) 0), &newval, false);

          //If Lockbit was already free return
          if (!TableEntry<ADD, REM>::getBits(Lindex[0], Lindex[1], oldval)) {
              return true;
          }


          uint64_cu res = atomicCAS(TableEntry<ADD, REM>::getValPtr(), oldval, newval);

          //Check if lockbit is now not set
          if (res == oldval) {
              return true;
          }
        }
    }

    __host__ __device__
    void print() {
        printf("%" PRIu64  "\n", TableEntry<ADD, REM>::val);
        return;
    }

    __host__ __device__
    ClearyEntry<ADD, REM> compareAndSwap(ClearyEntry<ADD, REM> comp, ClearyEntry<ADD, REM> swap) {
        uint64_cu newVal = atomicCAS(TableEntry<ADD, REM>::getValPtr(), comp.getValue(), swap.getValue());
        return ClearyEntry(newVal);
    }

};
