#include "hip/hip_runtime.h"
#ifndef ENTRYINCLUDED
#define ENTRYINCLUDED
#include "TableEntry.h"
#endif


template <class ADD, class REM>
class ClearyEntry : TableEntry <ADD, REM> {

private:
    int Rindex[2] = {  1, 56 };
    int Oindex[2] = { 57, 57 };
    int Vindex[2] = { 58, 58 };
    int Cindex[2] = { 59, 59 };
    int Aindex[2] = { 60, 62 };
    int Lindex[2] = { 63, 63 };


public:
    __host__ __device__
    ClearyEntry(ADD R, bool O, bool V, bool C, int A, bool L) {
        val = 0;
        setR(R);
        setO(O);
        setV(V);
        setC(C);
        setA(A);
        setL(L);
    }

    __host__ __device__
    ClearyEntry() {
        ClearyEntry(0, false, false, true, 0, false);
    }

    __host__ __device__
    void exchValue(ClearyEntry* x) {
        //Atomically set this value to the new one
        uint64_t old = atomicExch(&val, x->getValue());
        //Return an entry with prev val
        x->setValue(old);
        return;
    }

    __host__ __device__
    void setR(REM x) {
        setBits(Rindex[0], Rindex[1], x);
    }

    __host__ __device__
    REM getR() {
        return (REM)getBits(Rindex[0], Rindex[1]);
    }

    __host__ __device__
    void setO(bool x) {
        setBits(Oindex[0], Oindex[1], x);
    }

    __host__ __device__
    bool getO() {
        return (bool)getBits(Oindex[0], Oindex[1]);
    }

    __host__ __device__
    void setV(bool x) {
        setBits(Vindex[0], Vindex[1], x);
    }

    __host__ __device__
    bool getV() {
        return (bool)getBits(Vindex[0], Vindex[1]);
    }

    __host__ __device__
    void setC(bool x) {
        setBits(Cindex[0], Cindex[1], x);
    }

    __host__ __device__
    bool getC() {
        return (bool)getBits(Cindex[0], Cindex[1]);
    }

    __host__ __device__
    void setA(int x) {
        setBits(Aindex[0], Aindex[1], signed_to_unsigned(x, Aindex[1]-Aindex[0]));
    }

    __host__ __device__
    int getA() {
        return unsigned_to_signed(getBits(Aindex[0], Aindex[1]), Aindex[1] - Aindex[0]);
    }

    __host__ __device__
    void setL(bool x) {
        setBits(Lindex[0], Lindex[1], x);
    }

    __host__ __device__
    bool getL() {
        return getBits(Lindex[0], Lindex[1]);
    }

    __host__ __device__
    void print() {
        std::cout << std::bitset<64>(val) << "\n";
    }

};